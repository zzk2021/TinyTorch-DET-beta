#include "hip/hip_runtime.h"
/*
 * TinyTorch
 * @author 	: keith@robot9.me
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#if CUDA_VERSION >= 12010
#include <hip/hip_fp8.h>
#endif

#include <cassert>
#include <cfloat>
#include <iostream>
#include "TensorImpl_cpu.h"
#include "TensorImpl_cuda.cuh"
#include "TensorImpl_cuda.inc"

namespace TinyTorch {

const char* curandGetErrorString(hiprandStatus_t status);
const char* cublasGetErrorString(hipblasStatus_t status);

#define DTYPE_CASE(dtype_enum, cuda_type, dtype)                              \
    case dtype_enum: dtype = cuda_type; break;                                \

//
#define DTYPE_SWITCH(dtype_var, dtype)                                        \
    switch (dtype_var) {                                                      \
        DTYPE_CASE(Dtype::float32,   HIP_R_32F, dtype)                       \
        DTYPE_CASE(Dtype::bfloat16,  HIP_R_16BF, dtype)                      \
        DTYPE_CASE(Dtype::float16,   HIP_R_16F, dtype)                       \
        default: throw std::invalid_argument("Unsupported Dtype");            \
    }                                                                         \

#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA error in file '" << __FILE__ << "' in line "         \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

#define CURAND_CHECK(call)                                               \
  do {                                                                   \
    hiprandStatus_t err = call;                                           \
    if (err != HIPRAND_STATUS_SUCCESS) {                                  \
      std::cerr << "CURAND error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << curandGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUBLAS_CHECK(call)                                               \
  do {                                                                   \
    hipblasStatus_t err = call;                                           \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                  \
      std::cerr << "CUBLAS error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << cublasGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUDA_KERNEL_CHECK()                                                   \
  do {                                                                        \
    hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA kernel error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

static std::random_device _r;
unsigned long RandomGeneratorCUDA::seed_ = _r();
unsigned long RandomGeneratorCUDA::sequence_ = 0;

void* AllocatorCPU::allocatePinned(size_t size) {
  void* ptr = nullptr;
  CUDA_CHECK(hipHostMalloc(&ptr, size));
  return ptr;
}

void AllocatorCPU::deallocatePinned(void* ptr) {
  CUDA_CHECK(hipHostFree(ptr));
}

void AllocatorCUDA::allocate(void** ptr, size_t size) {
  CUDA_CHECK(hipMalloc(ptr, size));
}

void AllocatorCUDA::deallocate(void* ptr) {
  if (ptr) {
    CUDA_CHECK(hipFree(ptr));
  }
}

TensorOpsCUDA::TensorOpsCUDA(int32_t device, size_t blockSize)
    : cudaDeviceIdx_(device), blockSize_(blockSize) {
  CUDA_CHECK(hipSetDevice(cudaDeviceIdx_));
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp_, cudaDeviceIdx_));

  if (blockSize_ > deviceProp_.maxThreadsPerBlock) {
    blockSize_ = deviceProp_.maxThreadsPerBlock;
  }

  allocator_.setBaseAllocator(std::make_shared<AllocatorCUDA>());
}

TensorOpsCUDA::~TensorOpsCUDA() {
  allocator_.clear();
  if (blasHandle_) {
    hipblasDestroy(blasHandle_);
  }
}

hipblasHandle_t TensorOpsCUDA::getCublasHandle() {
  if (blasHandle_ == nullptr) {
    hipblasCreate(&blasHandle_);
  }
  return blasHandle_;
}

template <typename T>
TensorCudaCtx<T> TensorOpsCUDA::getTensorCtx(const TensorImpl& t) {
  TensorCudaCtx<T> ret{};
  ret.dimCount_ = t.dimCount_;
  ret.elemCount_ = t.elemCount_;
  memcpy(ret.shape_, t.shape_.data(), t.dimCount_ * sizeof(int32_t));
  memcpy(ret.strides_, t.strides_.data(), t.dimCount_ * sizeof(int32_t));
  ret.data_ = reinterpret_cast<T*>(t.data_);
  return ret;
}

template <typename OP>
void TensorOpsCUDA::opSingle_(TensorImpl& t) const {
  if (t.type_ == Dtype::float16)
    kSingleOp_<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(t.data_), t.elemCount_);
  if (t.type_ == Dtype::bfloat16)
    kSingleOp_<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*> (t.data_) , t.elemCount_);
  else
    kSingleOp_<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
       t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opSingle(const TensorImpl& t) const {
  auto result = TensorImpl::shape(t.shape(), t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    kSingleOp<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(t.data_), t.elemCount_);
  if (t.type_ == Dtype::bfloat16)
    kSingleOp<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_),reinterpret_cast<__hip_bfloat16*> (t.data_) , t.elemCount_);
  else
    kSingleOp<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      result.data_, t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a,
                                 const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_, a.type_);
  if (a.type() == Dtype::float32)
    kPairOp<OP, float><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  else if (a.type() == Dtype::bfloat16){
    kPairOp<OP, __hip_bfloat16><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      reinterpret_cast<__hip_bfloat16*>(b.data_), result.elemCount_);
  }
  else if (a.type() == Dtype::float16){
    kPairOp<OP, half><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      reinterpret_cast<half*>(b.data_), result.elemCount_);
  }
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a, float b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_, a.type_);

  if (a.type() == Dtype::float32)
    kPairScalarSecondOp<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b, a.elemCount_);

  else if (a.type() == Dtype::bfloat16)
    kPairScalarSecondOp<OP, __hip_bfloat16><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      static_cast<__hip_bfloat16>(b), a.elemCount_);

  else if (a.type() == Dtype::float16)
    kPairScalarSecondOp<OP, half><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      static_cast<half>(b), a.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(float a, const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_, b.type_);

  if (b.type() == Dtype::float32)
    kPairScalarFirstOp<OP, float><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      result.data_, a, b.data_, b.elemCount_);

  else if (b.type() == Dtype::bfloat16)
    kPairScalarFirstOp<OP, __hip_bfloat16><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_),
       static_cast<__hip_bfloat16>(a),
      reinterpret_cast<__hip_bfloat16*>(b.data_), b.elemCount_);

  else if (b.type() == Dtype::float16)
    kPairScalarFirstOp<OP, half><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_),
      static_cast<half>(a),
      reinterpret_cast<half*>(b.data_), b.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarFirst(const TensorImpl& a,
                                            const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_, b.type_);

  if (b.type() == Dtype::float32)
    kPairScalarFirstOp<OP, float><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);

  else if (b.type() == Dtype::bfloat16)
    kPairScalarFirstOp<OP, __hip_bfloat16><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      reinterpret_cast<__hip_bfloat16*>(b.data_), result.elemCount_);

  else if (b.type() == Dtype::float16)
    kPairScalarFirstOp<OP, half><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      reinterpret_cast<half*>(b.data_), result.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarSecond(const TensorImpl& a,
                                             const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_);
  kPairScalarSecondOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, float b) const {

  if (t.type() == Dtype::float32)
    kPairScalarSecondOp_<OP, float><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      t.data_, b, t.elemCount_);

  else if (t.type() == Dtype::bfloat16)
    kPairScalarSecondOp_<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(t.data_), __float2bfloat16(b),
      t.elemCount_);

  else if (t.type() == Dtype::float16)
    kPairScalarSecondOp_<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(t.data_), __float2half(b),
      t.elemCount_);

  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, const TensorImpl& b) const {
  kPairOp_<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, b.data_,
                                                              t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPairScalarFirst_(TensorImpl& a,
                                       const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape_, b.device_);
  kPairScalarFirstOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  a = std::move(result);
}

template <typename OP>
void TensorOpsCUDA::opPairScalarSecond_(TensorImpl& a,
                                        const TensorImpl& b) const {
  kPairScalarSecondOp_<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP, typename T>
void TensorOpsCUDA::broadcastImpl(TensorImpl& result, const TensorImpl& a,
                                  const TensorImpl& b) const {
  // fast broadcast with a

  if (b.elemCount_ == result.elemCount_) {

    if (isLeadingOnes(a.shape())) {
        kBroadcastOpFast<OP, true, true, T>
              <<<getGridSize(result.elemCount_), getBlockSize()>>>(
                  reinterpret_cast<T*>(result.data_),
                  reinterpret_cast<T*>(a.data_),
                  reinterpret_cast<T*>(b.data_),
                  a.elemCount_, result.elemCount_);

      CUDA_KERNEL_CHECK();
      return;
    }
    if (isTrailingOnes(a.shape())) {
      kBroadcastOpFast<OP, false, true, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              result.elemCount_ / a.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  // fast broadcast with b
  if (a.elemCount_ == result.elemCount_) {
    if (isLeadingOnes(b.shape())) {
      kBroadcastOpFast<OP, true, false, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              b.elemCount_, result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }

    if (isTrailingOnes(b.shape())) {
      kBroadcastOpFast<OP, false, false, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              result.elemCount_ / b.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  const auto ctxA = getTensorCtx<T>(a);
  const auto ctxB = getTensorCtx<T>(b);
  const auto ctxC = getTensorCtx<T>(result);
  kBroadcastOpCommon<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      ctxC, ctxA, ctxB, result.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairBroadcast(const TensorImpl& a,
                                          const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return {};
  }

  if (comp == ShapeCompatible_SameShape) {
    return opPair<OP>(a, b);
  }

  auto result = TensorImpl::shape(retShape, a.device_);
  broadcastImpl<OP>(result, a, b);
  return result;
}



template <typename OP>
void TensorOpsCUDA::opPairBroadcast_(TensorImpl& a, const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return;
  }
  if (comp == ShapeCompatible_SameShape) {
    opPair_<OP>(a, b);
    return;
  }
  auto result = TensorImpl::shape(retShape, a.device_, a.type_);

  if (a.type_ == Dtype::float32)
    broadcastImpl<OP>(result, a, b);
  else if (a.type_ == Dtype::float16)
    broadcastImpl<OP, half>(result, a, b);
  else if (a.type_ == Dtype::bfloat16)
    broadcastImpl<OP, __hip_bfloat16>(result, a, b);

  a = std::move(result);
}

template <typename OP ,typename T>
void TensorOpsCUDA::reduceAllImpl(T* dOutput, const T* dInput,
                                  int32_t n, int32_t m, KernelFunc<OP, T> kernel) {
  auto blocks = getGridSize(n);

  T* dTmp = nullptr;
  allocate(reinterpret_cast<void**>(&dTmp), m * blocks * sizeof(T));

  kernel<<<m * blocks, getBlockSize()>>>(dTmp, dInput, n, m);
  CUDA_KERNEL_CHECK();
  while (blocks > 1) {
    auto currBlocks = blocks;
    blocks = getGridSize(currBlocks);
    kReduceAll<OP><<<m * blocks, getBlockSize()>>>(dTmp, dTmp, currBlocks, m);
    CUDA_KERNEL_CHECK();
  }
  copyOnDevice(dOutput, dTmp, m * sizeof(T));
  deallocate(dTmp);
}

template <typename OP, typename T>
void TensorOpsCUDA::reduceAll(T* dOutput, const T* dInput, int32_t n,
                              int32_t m) {
  reduceAllImpl<OP, T>(dOutput, dInput, n, m, kReduceAll<OP, T>);
}

template <typename OP>
void TensorOpsCUDA::reduceAllIdx(float* dOutput, const float* dInput, int32_t n,
                                 int32_t m) {
  reduceAllImpl<OP>(dOutput, dInput, n, m, kReduceAllIdx<OP>);
}

template <typename OP, typename T>
void TensorOpsCUDA::reduceAllLastDim(T* dOutput, const T* dInput,
                                     int32_t n, int32_t m) {
  reduceAllImpl<OP, T>(dOutput, dInput, n, m, kReduceAllLastDim<OP, T>);

}

template <typename OP, typename T>
std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::reduceDim(const TensorImpl& t,
                                                           int32_t dim,
                                                           bool keepDims) {
  if (dim < 0) {
    dim += t.dimCount_;
  }
  if (dim < 0 || dim >= t.dimCount_) {
    error(__FUNCTION__, TensorError_InvalidAxis);
    return {};
  }

  const auto retShape = getReduceShape(t, dim, false);
  auto values = TensorImpl::shape(retShape, t.device_, t.type_);
  auto indices = TensorImpl::shape(retShape, t.device_);

  if (dim == t.dimCount_ - 1) {
    kReduceLastDim<OP, T><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        reinterpret_cast<T*>(values.data_), indices.data_, reinterpret_cast<T*>(t.data_),
        t.shape_[dim], values.elemCount_);
  } else {
    auto ctxT = getTensorCtx<T>(t);
    auto ctxValues = getTensorCtx<T>(values);
    kReduceDim<OP, T><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        ctxValues, indices.data_, ctxT, dim, ctxValues.elemCount_);
  }
  CUDA_KERNEL_CHECK();

  if (keepDims) {
    const auto shapeKeepDims = getReduceShape(t, dim, true);
    values.reshape_(shapeKeepDims);
    indices.reshape_(shapeKeepDims);
  }

  // values as the same type as origin data
  // but indices is float32
  return {values, indices};
}

template <typename T>
void TensorOpsCUDA::transpose2D(T* out, const T* in, int32_t width,
                                int32_t height) {
  dim3 blockSize(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_DIM);
  dim3 gridSize((width + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM,
                (height + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM);
  kTranspose<<<gridSize, blockSize>>>(out, in, width, height);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::allocate(void** ptr, size_t size) {
  allocator_.allocate(ptr, size);
}

void TensorOpsCUDA::deallocate(void* ptr) { allocator_.deallocate(ptr); }

void TensorOpsCUDA::copyHostToDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
}

void TensorOpsCUDA::convertTypeOnDevice(void* dst, void* src, size_t count, Dtype Ti ,Dtype To) {
   // src = static_cast<float*>(src);
    size_t threads_per_block = 256;
    size_t blocks = (count + threads_per_block - 1) / threads_per_block;
    if (Ti == Dtype::float32) {
        assert(static_cast<float*>(src) != nullptr);
    } else if (Ti == Dtype::bfloat16) {
        assert(static_cast<__hip_bfloat16*>(src) != nullptr);
    } else if (Ti == Dtype::float16) {
        assert(static_cast<half*>(src) != nullptr);
    }

    if (To == Dtype::float32) {
        assert(static_cast<float*>(dst) != nullptr);
    } else if (To == Dtype::bfloat16) {
        assert(static_cast<__hip_bfloat16*>(dst) != nullptr);
    } else if (To == Dtype::float16) {
        assert(static_cast<half*>(dst) != nullptr);
    }

    if (Ti == Dtype::float32 && To == Dtype::float16) {
        convertFloatToHalfKernel<<<blocks, threads_per_block>>>(
            static_cast<float*>(src), static_cast<half*>(dst), count);
    } else if (Ti == Dtype::float32 && To == Dtype::bfloat16) {
        convertFloatToBf16Kernel<<<blocks, threads_per_block>>>(
            static_cast<float*>(src), static_cast<__hip_bfloat16*>(dst), count);
    } else if (Ti == Dtype::bfloat16 && To == Dtype::float32) {
        convertBf16ToFloatKernel<<<blocks, threads_per_block>>>(
            static_cast<__hip_bfloat16*>(src), static_cast<float*>(dst), count);
    } else if (Ti == Dtype::float16 && To == Dtype::float32) {
        convertHalfToFloatKernel<<<blocks, threads_per_block>>>(
            static_cast<half*>(src), static_cast<float*>(dst), count);
    } else if (Ti == To) {
        if (Ti == Dtype::float32)
            hipMemcpy(dst, src, count * sizeof(float), hipMemcpyDeviceToDevice);
        else if (Ti == Dtype::bfloat16 || Ti == Dtype::float16)
            hipMemcpy(dst, src, count * sizeof(half), hipMemcpyDeviceToDevice);
    } else {
        LOGW("Type conversion from %d to %d is not supported, keeping the same type",
             Ti, To);
        //
    }
    CUDA_KERNEL_CHECK();
}



void TensorOpsCUDA::copyOnDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice));
}

void TensorOpsCUDA::copyDeviceToHost(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
}

void TensorOpsCUDA::fillConstant_(float* dst, float val, size_t count) {
  kFillConstant<<<getGridSize(count, 4), getBlockSize()>>>(dst, val, count);
  CUDA_KERNEL_CHECK();
}


void TensorOpsCUDA::fillConstant_(TensorImpl& t, float val) {

  if(t.type_ == Dtype::float16)
    kFillConstant<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(reinterpret_cast<half*>(t.data_),
                                                                    __float2half(val),
                                                                  t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    kFillConstant<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(reinterpret_cast<__hip_bfloat16*>(t.data_),
                                                                    __float2bfloat16(val),
                                                                  t.elemCount_);
  else
    kFillConstant<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(t.data_, val,
                                                                  t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillLinSpace_(float* dst, float start, float step,
                                  size_t count) {
  kFillLinSpace<<<getGridSize(count, 4), getBlockSize()>>>(dst, start, step,
                                                           count);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandUniform_(TensorImpl& t, float min, float max) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandUniform<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, min, max, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandNormal_(TensorImpl& t) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandNormal<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, 0.f, 1.f, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandBernoulli_(TensorImpl& t, float p) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandBernoulli<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, p, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaAdd>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaAdd>(a, b);
  }
  return opPairBroadcast<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaSub>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaSub>(a, b);
  }
  return opPairBroadcast<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaMul>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMul>(a, b);
  }
  return opPairBroadcast<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaDiv>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaDiv>(a, b);
  }
  return opPairBroadcast<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaPow>(a, b);
  }
  return opPairBroadcast<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const float& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const float& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const float& b) {
  return opPair<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const float& b) {
  return opPair<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const float& a, const TensorImpl& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const float& a, const TensorImpl& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const float& a, const TensorImpl& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const float& a, const TensorImpl& b) {
  return opPair<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaAdd>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaAdd>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaSub>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaSub>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaMul>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaMul>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaDiv>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaDiv>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const float& b) {
  opPair_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const float& b) {
  opPair_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const float& b) {
  opPair_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const float& b) {
  opPair_<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaEq>(a, b);
  }
  return opPairBroadcast<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaNe>(a, b);
  }
  return opPairBroadcast<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGe>(a, b);
  }
  return opPairBroadcast<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGt>(a, b);
  }
  return opPairBroadcast<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLe>(a, b);
  }
  return opPairBroadcast<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLt>(a, b);
  }
  return opPairBroadcast<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMax>(a, b);
  }
  return opPairBroadcast<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMin>(a, b);
  }
  return opPairBroadcast<OpCudaMin>(a, b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const float& b) {
  return opPair<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const float& b) {
  return opPair<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMin>(a, b);
}

void TensorOpsCUDA::abs_(TensorImpl& t) { opSingle_<OpCudaSin_>(t); }

void TensorOpsCUDA::sin_(TensorImpl& t) { opSingle_<OpCudaSin_>(t); }

void TensorOpsCUDA::cos_(TensorImpl& t) { opSingle_<OpCudaCos_>(t); }

void TensorOpsCUDA::sqrt_(TensorImpl& t) { opSingle_<OpCudaSqrt_>(t); }

void TensorOpsCUDA::tanh_(TensorImpl& t) { opSingle_<OpCudaTanh_>(t); }

void TensorOpsCUDA::exp_(TensorImpl& t) { opSingle_<OpCudaExp_>(t); }

void TensorOpsCUDA::log_(TensorImpl& t) { opSingle_<OpCudaLog_>(t); }

TensorImpl TensorOpsCUDA::sin(const TensorImpl& t) {
  return opSingle<OpCudaSin>(t);
}

TensorImpl TensorOpsCUDA::cos(const TensorImpl& t) {
  return opSingle<OpCudaCos>(t);
}

TensorImpl TensorOpsCUDA::abs(const TensorImpl& t) {
  return opSingle<OpCudaAbs>(t);
}

TensorImpl TensorOpsCUDA::sqrt(const TensorImpl& t) {
  return opSingle<OpCudaSqrt>(t);
}

TensorImpl TensorOpsCUDA::tanh(const TensorImpl& t) {
  return opSingle<OpCudaTanh>(t);
}

TensorImpl TensorOpsCUDA::exp(const TensorImpl& t) {
  return opSingle<OpCudaExp>(t);
}

TensorImpl TensorOpsCUDA::log(const TensorImpl& t) {
  return opSingle<OpCudaLog>(t);
}

void TensorOpsCUDA::clampMin_(TensorImpl& t, float min) {
  opPair_<OpCudaMax>(t, min);
}

void TensorOpsCUDA::clampMax_(TensorImpl& t, float max) {
  opPair_<OpCudaMin>(t, max);
}

void TensorOpsCUDA::clamp_(TensorImpl& t, float min, float max) {
  kClamp_<<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, min, max,
                                                         t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::clampMin(const TensorImpl& t, float min) {
  return opPair<OpCudaMax>(t, min);
}

TensorImpl TensorOpsCUDA::clampMax(const TensorImpl& t, float max) {
  return opPair<OpCudaMin>(t, max);
}

TensorImpl TensorOpsCUDA::clamp(const TensorImpl& t, float min, float max) {
  auto ret = TensorImpl::shape(t.shape_, t.device_);
  kClamp<<<getGridSize(t.elemCount_), getBlockSize()>>>(ret.data_, t.data_, min,
                                                        max, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::min(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAll<OpCudaReduceMin>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::max(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceMax>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceMax>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceMax>(ret.data_, t.data_, t.elemCount_);
  return ret;
}


TensorImpl TensorOpsCUDA::sum(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceSum>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceSum>(ret.data_, t.data_, t.elemCount_);

  const auto r = 1.f / static_cast<float>(t.elemCount_);
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::var(const TensorImpl& t, bool unbiased) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  const auto meanVal = mean(t);
  const auto squaredDiff = TensorImpl::shape({t.elemCount_}, t.device_);
  kSquaredDiff<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      squaredDiff.data_, t.data_, meanVal.data_, t.elemCount_);

  auto ret = TensorImpl::scalar(t.device_);
  reduceAll<OpCudaReduceSum>(ret.data_, squaredDiff.data_, t.elemCount_);

  const auto n = static_cast<float>(t.elemCount_);
  auto r = 1.f / n;
  if (unbiased) {
    r *= n / (n - 1.f);
  }
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::argmin(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAllIdx<OpCudaReduceMin>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::argmax(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAllIdx<OpCudaReduceMax>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::min(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_)};
  }
  return reduceDim<OpCudaReduceMin>(t, dim, keepDims);
}



std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::max(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_, t.type_)};
  }
  if (t.type_ == Dtype::float16)
    return reduceDim<OpCudaReduceMax, half>(t, dim, keepDims);
  else if(t.type_ == Dtype::bfloat16)
    return reduceDim<OpCudaReduceMax, __hip_bfloat16>(t, dim, keepDims);
  else
    return reduceDim<OpCudaReduceMax>(t, dim, keepDims);
}

TensorImpl TensorOpsCUDA::sum(const TensorImpl& t,
                              const std::vector<int32_t>& dims, bool keepDims) {
  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }
  if (t.dimCount_ == 0) {
    return t;
  }

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto ret = TensorImpl::shape(retShape, t.device_, t.type_);

  if (dims.size() == 1) {
    auto d = dims[0];
    if (d < 0) {
      d += t.dimCount_;
    }

    // first dim
    if (d == 0) {
      const auto dimSize = t.shape_.front();
      const auto tmp = TensorImpl::shape(t.shape_, t.device_, t.type_);
      if (t.type_ == Dtype::float16)
        transpose2D(reinterpret_cast<half*>(tmp.data_),
                    reinterpret_cast<half*>(t.data_),
                    ret.elemCount_,
                    dimSize);
      else if (t.type_ == Dtype::bfloat16)
        transpose2D(reinterpret_cast<__hip_bfloat16*>(tmp.data_),
                    reinterpret_cast<__hip_bfloat16*>(t.data_),
                    ret.elemCount_,
                    dimSize);
      else
        transpose2D(tmp.data_, t.data_, ret.elemCount_, dimSize);


      if (t.type_ == Dtype::float16)
        reduceAllLastDim<OpCudaReduceSum, half>(reinterpret_cast<half*>(ret.data_),
                    reinterpret_cast<half*>(tmp.data_),
                    dimSize,
                    ret.elemCount_);
      else if (t.type_ == Dtype::bfloat16)
        reduceAllLastDim<OpCudaReduceSum, __hip_bfloat16>(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                    reinterpret_cast<__hip_bfloat16*>(tmp.data_),
                    dimSize,
                    ret.elemCount_);
      else
        reduceAllLastDim<OpCudaReduceSum, float>(ret.data_,
                    tmp.data_,
                    dimSize,
                    ret.elemCount_);
      return ret;
    }

    // last dim
    if (d == t.dimCount_ - 1) {
      const auto dimSize = t.shape_.back();
      if (t.type_ == Dtype::float16)
        reduceAllLastDim<OpCudaReduceSum, half>(reinterpret_cast<half*>(ret.data_),
                    reinterpret_cast<half*>(t.data_),
                    dimSize,
                    ret.elemCount_);
      else if (t.type_ == Dtype::bfloat16)
        reduceAllLastDim<OpCudaReduceSum, __hip_bfloat16>(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                    reinterpret_cast<__hip_bfloat16*>(t.data_),
                    dimSize,
                    ret.elemCount_);
      else
        reduceAllLastDim<OpCudaReduceSum, float>(ret.data_,
                    t.data_,
                    dimSize,
                    ret.elemCount_);
      return ret;
    }
  }

    auto ctxT = getTensorCtx(t);
    fillConstant_(ret, 0);
    kReduceSum<<<getGridSize(t.elemCount_), getBlockSize()>>>(
    ret.data_, ctxT, inAxis, t.elemCount_);

  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t,
                               const std::vector<int32_t>& dims,
                               bool keepDims) {
  auto ret = sum(t, dims, keepDims);
  if (!ret.empty()) {
    auto reduceSize = (float)t.elemCount_ / (float)ret.elemCount_;
    auto r = 1.f / reduceSize;
    mul_(ret, r);
  }
  return ret;
}

TensorImpl TensorOpsCUDA::var(const TensorImpl& t,
                              const std::vector<int32_t>& dims, bool unbiased,
                              bool keepDims) {
  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto ret = TensorImpl::shape(retShape, t.device_);

  auto meanTensor = mean(t, dims, true);
  fillConstant_(ret, 0);

  auto ctxT = getTensorCtx(t);
  kReduceVar<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ret.data_, ctxT, meanTensor.data_, inAxis, t.elemCount_);
  CUDA_KERNEL_CHECK();

  auto reduceSize = (float)t.elemCount_ / (float)ret.elemCount_;
  auto r = 1.f / reduceSize;
  if (unbiased) {
    r *= reduceSize / (reduceSize - 1.f);
  }
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::permute(const TensorImpl& t,
                                  const std::vector<int32_t>& dims) {
  auto retShape = t.shape_;
  reorderIndices(retShape.data(), dims);
  auto ret = TensorImpl::shape(retShape, t.device_, t.type_);

  auto ctxT = getTensorCtx(t);
  auto ctxRet = getTensorCtx(ret);

  auto* dimsDataPtr = (FixedVector<int32_t>*)dims.data();
  kPermute<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ctxRet, ctxT, *dimsDataPtr, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}



TensorImpl TensorOpsCUDA::transpose2D(const TensorImpl& t) {
  auto ret = TensorImpl::shape({t.shape_[1], t.shape_[0]}, t.device_, t.type_);
  if (t.type_ == Dtype::float16)
      transpose2D(reinterpret_cast<half*>(ret.data_),
                  reinterpret_cast<half*>(t.data_), t.shape_[1], t.shape_[0]);
  else if (t.type_ == Dtype::bfloat16)
      transpose2D(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                  reinterpret_cast<__hip_bfloat16*>(t.data_), t.shape_[1], t.shape_[0]);
  else
      transpose2D(ret.data_, t.data_, t.shape_[1], t.shape_[0]);
  return ret;
}

TensorImpl TensorOpsCUDA::index(
    const TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  Shape retShape = {fistDim};
  for (auto i = len; i < t.dimCount_; i++) {
    retShape.push_back(t.shape_[i]);
  }
  auto retTensor = TensorImpl::shape(retShape, t.device_);

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndex<<<getGridSize(fistDim), getBlockSize()>>>(
      retTensor.data_, ctxT, indicesData, dimStride, len, fistDim);
  CUDA_KERNEL_CHECK();
  return retTensor;
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices, float val) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(fistDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val, fistDim);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices,
    const TensorImpl& val) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  assert(val.elemCount_ == dimStride * fistDim);

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(fistDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val.data_, fistDim);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::im2col(const TensorImpl& t, Size2D kernel,
                                 Size2D stride, Size2D padding) {
  // this: [C, H, W], [N, C, H, W]
  assert(t.dimCount_ == 3 || t.dimCount_ == 4);
  int32_t batch = (t.dimCount_ == 4) ? t.shape_[0] : 1;
  int32_t channels = (t.dimCount_ == 4) ? t.shape_[1] : t.shape_[0];
  int32_t height = (t.dimCount_ == 4) ? t.shape_[2] : t.shape_[1];
  int32_t width = (t.dimCount_ == 4) ? t.shape_[3] : t.shape_[2];
  int32_t outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  int32_t outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  int32_t colH = outH * outW;
  int32_t colW = channels * kernel.h * kernel.w;
  auto retTensor = TensorImpl::shape({batch * colH, colW}, t.device_, t.type_);

  int32_t imStride = t.strides_[0];
  int totalElements = batch * outH * outW * channels * kernel.h * kernel.w;

  if (t.type_ == Dtype::float32)
    kIm2Col<<<getGridSize(totalElements), getBlockSize()>>>(
      retTensor.data_, t.data_, batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
  else if (t.type_ == Dtype::float16){
    kIm2Col<half><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<half*>(retTensor.data_), reinterpret_cast<half*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
    }
  else if (t.type_ == Dtype::bfloat16){
    kIm2Col<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(retTensor.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
    }

  CUDA_KERNEL_CHECK();
  return retTensor;
}
TensorImpl TensorOpsCUDA::im2col1D(const TensorImpl& t,
                                 Size1D kernel_size,
                                 Size1D stride,
                                 Size1D padding) {

  assert(t.dimCount_ == 2 || t.dimCount_ == 3);

  const int32_t batch = (t.dimCount_ == 3) ? t.shape_[0] : 1;
  const int32_t channels = (t.dimCount_ == 3) ? t.shape_[1] : t.shape_[0];
  const int32_t length = (t.dimCount_ == 3) ? t.shape_[2] : t.shape_[1];

  const int32_t outLength = (length - kernel_size.d + 2 * padding.d) / stride.d + 1;

  const int32_t colH = outLength;
  const int32_t colW = channels * kernel_size.d;
  auto retTensor = TensorImpl::shape({batch * colH, colW}, t.device_, t.type_);

  const int totalElements = batch * outLength * channels * kernel_size.d;
  const int imStride = (t.dimCount_ == 3) ? t.strides_[2] : t.strides_[1];

  if (t.type_ == Dtype::float32) {
    kIm2Col1D<<<getGridSize(totalElements), getBlockSize()>>>(
        retTensor.data(),
        t.data(),
        batch,
        channels,
        length,
        outLength,
        kernel_size.d,
        stride.d,
        padding.d,
        imStride,
        colH,
        colW
    );
  }
  else if (t.type_ == Dtype::float16) {
    kIm2Col1D<half><<<getGridSize(totalElements), getBlockSize()>>>(
        reinterpret_cast<half*>(retTensor.data()),
        reinterpret_cast<const half*>(t.data()),
        batch,
        channels,
        length,
        outLength,
        kernel_size.d,
        stride.d,
        padding.d,
        imStride,
        colH,
        colW
    );
  }
  else if (t.type_ == Dtype::bfloat16) {
    kIm2Col1D<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
        reinterpret_cast<__hip_bfloat16*>(retTensor.data()),
        reinterpret_cast<const __hip_bfloat16*>(t.data()),
        batch,
        channels,
        length,
        outLength,
        kernel_size.d,
        stride.d,
        padding.d,
        imStride,
        colH,
        colW
    );
  }

  CUDA_KERNEL_CHECK();
  return retTensor;
}

TensorImpl TensorOpsCUDA::col2im(const TensorImpl& t, const Shape& shape,
                                 Size2D kernel, Size2D stride, Size2D padding) {
  // shape: [C, H, W], [N, C, H, W]
  assert(shape.size() == 3 || shape.size() == 4);
  int32_t batch = (shape.size() == 4) ? shape[0] : 1;
  int32_t channels = (shape.size() == 4) ? shape[1] : shape[0];
  int32_t height = (shape.size() == 4) ? shape[2] : shape[1];
  int32_t width = (shape.size() == 4) ? shape[3] : shape[2];

  auto outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  auto outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  // int32_t colH = outH * outW;
  int32_t colW = channels * kernel.h * kernel.w;

  auto retTensor = TensorImpl::zeros(shape, t.device_, t.type_);

  auto imStride = retTensor.strides_[0];
  int totalElements = batch * channels * height * width;

  if (t.type_ == Dtype::float32)
    kCol2Im<<<getGridSize(totalElements), getBlockSize()>>>(
      retTensor.data_, t.data_, batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
  else if (t.type_ == Dtype::float16){

    kCol2Im<half><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<half*>(retTensor.data_), reinterpret_cast<half*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
    }
  else if (t.type_ == Dtype::bfloat16){
    kCol2Im<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(retTensor.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
    }
  CUDA_KERNEL_CHECK();
  return retTensor;
}

TensorImpl TensorOpsCUDA::col2im1D(const TensorImpl& t,
                                 const Shape& shape,
                                 Size1D kernel,
                                 Size1D stride,
                                 Size1D padding) {
    assert(shape.size() == 2 || shape.size() == 3);

    const int32_t batch = (shape.size() == 3) ? shape[0] : 1;
    const int32_t channels = (shape.size() == 3) ? shape[1] : shape[0];
    const int32_t length = (shape.size() == 3) ? shape[2] : shape[1];

    const int32_t outLength = (length - kernel.d + 2 * padding.d) / stride.d + 1;

    const int32_t colW = channels * kernel.d;

    auto retTensor = TensorImpl::zeros(shape, t.device_, t.type_);

    const int32_t imStride = retTensor.strides_.back();
    const int totalElements = batch * channels * length;

    if (t.type_ == Dtype::float32) {
        kCol2Im1D<float><<<getGridSize(totalElements), getBlockSize()>>>(
            retTensor.data(),
            t.data(),
            batch,
            channels,
            length,
            outLength,
            kernel.d,
            stride.d,
            padding.d,
            colW
        );
    }
    else if (t.type_ == Dtype::float16) {
        kCol2Im1D<half><<<getGridSize(totalElements), getBlockSize()>>>(
            reinterpret_cast<half*>(retTensor.data()),
            reinterpret_cast<const half*>(t.data()),
            batch,
            channels,
            length,
            outLength,
            kernel.d,
            stride.d,
            padding.d,
            colW
        );
    }
    else if (t.type_ == Dtype::bfloat16) {
        kCol2Im1D<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
            reinterpret_cast<__hip_bfloat16*>(retTensor.data()),
            reinterpret_cast<const __hip_bfloat16*>(t.data()),
            batch,
            channels,
            length,
            outLength,
            kernel.d,
            stride.d,
            padding.d,
            colW
        );
    }
    CUDA_KERNEL_CHECK();
    return retTensor;
}

TensorImpl TensorOpsCUDA::dot(const TensorImpl& a, const TensorImpl& b) {
  auto ret = TensorImpl::scalar(0.f, a.device_);
  auto sharedMemSize = getBlockSize() * sizeof(float);
  kDot<<<getGridSize(a.elemCount_), getBlockSize(), sharedMemSize>>>(
      ret.data_, a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::triangle(const TensorImpl& t, int32_t diagonal,
                                    bool lower) {
  auto ret = TensorImpl::shape(t.shape_, t.device_);
  const auto rows = t.shape_[0];
  const auto cols = t.shape_[1];

  dim3 blockSize(WARP_SIZE, WARP_SIZE);
  dim3 gridSize((cols + blockSize.x - 1) / blockSize.x,
                (rows + blockSize.y - 1) / blockSize.y);

  if (lower) {
    kTriangle<true>
        <<<gridSize, blockSize>>>(ret.data_, t.data_, rows, cols, diagonal);
  } else {
    kTriangle<false>
        <<<gridSize, blockSize>>>(ret.data_, t.data_, rows, cols, diagonal);
  }
  CUDA_KERNEL_CHECK();
  return ret;
}

void TensorOpsCUDA::gemm(float* c, const float* a, const float* b, int32_t m,
                         int32_t k, int32_t n, bool transA, bool transB,const  Dtype Ta,
                         Dtype Tc) {
  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  int lda = transA ? m : k;
  int ldb = transB ? k : n;
  int ldc = n;

  const float alpha = 1.f;
  const float beta = 0.f;
  if (Ta == Dtype::float32)
    CUBLAS_CHECK(hipblasSgemm(getCublasHandle(), opB, opA, n, m, k, &alpha, b, ldb,
                           a, lda, &beta, c, ldc));
  else{
    hipDataType dtypea;
    DTYPE_SWITCH(Ta, dtypea);
    hipDataType dtypec;
    DTYPE_SWITCH(Tc, dtypec);
      CUBLAS_CHECK(hipblasGemmEx(
        getCublasHandle(),
        opB, opA,
        n, m, k,
        &alpha,
        b, dtypea, ldb,
        a, dtypea, lda,
        &beta,
        c, dtypec, ldc,
        HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
    ));
  }
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::split(
    const TensorImpl& input,
    int32_t split_size0,
    int32_t split_size1,
    int32_t dim)
{
  Shape input_shape = input.shape();

  std::vector<int32_t> output_shape0 = input.shape();
  std::vector<int32_t> output_shape1 = input.shape();
  output_shape0[dim] = split_size0;
  output_shape1[dim] = split_size1;
  auto ret0 = TensorImpl::shape(output_shape0, input.device_, input.type_);
  auto ret1 = TensorImpl::shape(output_shape1, input.device_, input.type_);

  int32_t threads_per_block = 256;
  int32_t total_elems = input.numel();
  int32_t blocks = (total_elems + threads_per_block - 1) / threads_per_block;

  if (input.type() ==  Dtype::float32)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
        input.data(),
        ret0.data_,
        ret1.data_,
        input.shape().data(),
        split_size0,
        split_size1, dim,total_elems
    );
  else if (input.type() ==  Dtype::float16)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
       reinterpret_cast<const half*>(input.data()),
       reinterpret_cast<half*>(ret0.data_),
       reinterpret_cast<half*>(ret1.data_),
       input.shape().data(),
        split_size0,
        split_size1,
       dim, total_elems
   );
  else if (input.type() ==  Dtype::bfloat16)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
       reinterpret_cast<const __hip_bfloat16*>(input.data()),
       reinterpret_cast<__hip_bfloat16*>(ret0.data()),
       reinterpret_cast<__hip_bfloat16*>(ret1.data()),
       input.shape().data(),
        split_size0,
        split_size1,
       dim, total_elems
   );
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return {ret0, ret1};
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::leakyrelu(const TensorImpl& a, float rate){
  int32_t threads_per_block = 256;
  int32_t total_elems = a.numel();
  int32_t blocks = (total_elems + threads_per_block - 1) / threads_per_block;
  auto ret = TensorImpl::shape(a.shape_,a.device_,a.type_);
  auto mask = TensorImpl::shape(a.shape_, a.device_, Dtype::int8); // Mask 为布尔类型

  //auto ret = a * (a > 0.f) + a * (a <= 0.f) * rate;
  //return ret;
  if (a.type() == Dtype::float32)
       leaky_relu_kernel<float><<<blocks, threads_per_block>>>(
            a.data(),
            ret.data(),
            reinterpret_cast<bool*>(mask.data()),
            rate,
            total_elems
        );
  else if (a.type() == Dtype::float16)
      leaky_relu_kernel<half><<<blocks, threads_per_block>>>(
            reinterpret_cast<const half*>(a.data()),
            reinterpret_cast<half*>(ret.data()),
            reinterpret_cast<bool*>(mask.data()),
            rate,
            total_elems
        );
  else if (a.type() ==  Dtype::bfloat16)
        leaky_relu_kernel<__hip_bfloat16><<<blocks, threads_per_block>>>(
            reinterpret_cast<const __hip_bfloat16*>(a.data()),
            reinterpret_cast<__hip_bfloat16*>(ret.data()),
            reinterpret_cast<bool*>(mask.data()),
            rate,
            total_elems
        );
  CUDA_KERNEL_CHECK();
  return {ret,mask};
}


TensorImpl TensorOpsCUDA::leakyrelu_backward(const TensorImpl& a, const TensorImpl& mask, float rate){
  int32_t threads_per_block = 256;
  int32_t total_elems = a.numel();
  int32_t blocks = (total_elems + threads_per_block - 1) / threads_per_block;
  auto output = TensorImpl::shape(a.shape_,a.device_,a.type_);
  if (a.type() == Dtype::float32)
       leaky_relu_backward<float><<<blocks, threads_per_block>>>(
            a.data(),
            output.data(),
            reinterpret_cast<const bool*>(mask.data()),
            rate,
            total_elems
        );
  else if (a.type() == Dtype::float16)
      leaky_relu_backward<half><<<blocks, threads_per_block>>>(
            reinterpret_cast<const half*>(a.data()),
            reinterpret_cast<half*>(output.data()),
            reinterpret_cast<const bool*>(mask.data()),
            rate,
            total_elems
        );
  else if (a.type() ==  Dtype::bfloat16)
        leaky_relu_backward<__hip_bfloat16><<<blocks, threads_per_block>>>(
            reinterpret_cast<const __hip_bfloat16*>(a.data()),
            reinterpret_cast<__hip_bfloat16*>(output.data()),
            reinterpret_cast<const bool*>(mask.data()),
            rate,
            total_elems
        );
  CUDA_KERNEL_CHECK();
  return output;
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::from_mask(const TensorImpl& a, const TensorImpl& b) {
  assert(b.shape().size() <= a.shape().size());
  TensorImpl mask;
  if (a.shape() != b.shape()){
    for (int i = 0; i < a.shape().size(); ++i) {
      int dim_mask = (i < a.shape().size() - b.shape().size())
                         ? 1 : b.shape()[i - (a.shape().size() - b.shape().size())];
      int dim_target = a.shape()[i];
      if (dim_mask != 1 && dim_mask != dim_target) {
        assert(true);
      }
    }
    mask = TensorImpl::zerosLike(a,a.device(),a.type());
    if (a.type_ == Dtype::float32)
      broadcastImpl<OpCudaAssign>(mask, a, b);
  }else{
    mask = b;
  }
  int32_t ndim = a.shape_.size();
  int numElements = a.numel();
  int* d_prefixSum;
  allocate(reinterpret_cast<void**>(&d_prefixSum), numElements * sizeof(int));
  const int blockSize = 256;
  int gridSize = (numElements + blockSize - 1) / blockSize;

  computePrefixSumKernel<<<gridSize, blockSize>>>(
      mask.data(), d_prefixSum, numElements);
  thrust::device_ptr<int> thrust_prefixSum(d_prefixSum);
  thrust::inclusive_scan(thrust_prefixSum,
                         thrust_prefixSum + numElements, thrust_prefixSum);

  int totalValid;
  float *indice;
  copyDeviceToHost(&totalValid, d_prefixSum + numElements - 1, sizeof(int));
  allocate(reinterpret_cast<void**>(&indice), totalValid * sizeof(float));
  std::vector<float> indices_host;
  indices_host.resize(totalValid);

  TensorImpl ret = TensorImpl::shape({totalValid}, a.device(), a.type());

  gatherElementsKernel<<<gridSize, blockSize>>>(a.data(),
                                                d_prefixSum, indice, ret.data(), numElements);
  //scatterElementsKernel<<<gridSize, blockSize>>>(d_input, d_prefixSum, ret.data(), numElements);
  copyDeviceToHost(indices_host.data(), indice, totalValid * sizeof(float));

  deallocate(d_prefixSum);
  deallocate(indice);
  // Step 5: Check for kernel errors
  CUDA_KERNEL_CHECK();
  TensorImpl indices_t =  TensorImpl(indices_host,a.device());

  return {ret, indices_t};
}


TensorImpl TensorOpsCUDA::from_mask_backward(
    const TensorImpl& grad_output,
    const TensorImpl& indices,
    const std::vector<int32_t>& a_shape
) {
    TensorImpl grad_input = TensorImpl::zeros(a_shape, grad_output.device_, grad_output.type_);

    int totalValid = indices.numel();
    const int blockSize = 256;
    int gridSize = (totalValid + blockSize - 1) / blockSize;
    scatterGradKernel<<<gridSize, blockSize>>>(
        grad_output.data(),
        indices.data_,
        grad_input.data(),
        totalValid
    );
    CUDA_KERNEL_CHECK();
    return grad_input;
}

TensorImpl TensorOpsCUDA::from_slice(const TensorImpl& a, std::vector<int> starts, std::vector<int> ends) {
    int32_t ndim = a.shape_.size();

    // Step 1: Compute new shape
    std::vector<int> new_shape(ndim);

    for (int i = 0; i < ndim; ++i) {
        new_shape[i] = ends[i] - starts[i];
    }

    // Step 2: Create new tensor
    TensorImpl result = TensorImpl::shape(new_shape, a.device_);
    int new_size = result.numel();
    // Step 4: Launch kernel
    int threads_per_block = 256;
    int blocks = (new_size + threads_per_block - 1) / threads_per_block;

    int32_t *d_a_strides, *d_new_strides, *d_new_shape;
    int *d_starts;
    allocate(reinterpret_cast<void**>(&d_a_strides), ndim * sizeof(int32_t));
    allocate(reinterpret_cast<void**>(&d_starts), ndim * sizeof(int));
    allocate(reinterpret_cast<void**>(&d_new_strides), ndim * sizeof(int32_t));
    allocate(reinterpret_cast<void**>(&d_new_shape), ndim * sizeof(int32_t));

    copyHostToDevice(d_a_strides, a.strides_.data(), ndim * sizeof(int32_t));
    copyHostToDevice(d_starts, starts.data(), ndim * sizeof(int));
    copyHostToDevice(d_new_strides, result.strides_.data(), ndim * sizeof(int32_t));
    copyHostToDevice(d_new_shape, result.shape().data(), ndim * sizeof(int32_t));

    switch (ndim) {
        case 4:
            from_slice_kernel<4><<<blocks, threads_per_block>>>(
                    a.data_,                                    // a_data
                    d_a_strides,                          // a_strides
                    d_starts,                              // starts
                    d_new_strides,                     // new_strides
                    d_new_shape,                      // new_dim
                    result.data_,                               // result_data
                    new_size                                  // total_elements
                );
            break;
        case 5:
            from_slice_kernel<5><<<blocks, threads_per_block>>>(
                    a.data_,                                    // a_data
                    d_a_strides,                          // a_strides
                    d_starts,                              // starts
                    d_new_strides,                     // new_strides
                    d_new_shape,                      // new_dim
                    result.data_,                               // result_data
                    new_size                                  // total_elements
                );
            break;
        case 2:
          from_slice_kernel<2><<<blocks, threads_per_block>>>(
              a.data_,                                    // a_data
              d_a_strides,                          // a_strides
              d_starts,                              // starts
              d_new_strides,                     // new_strides
              d_new_shape,                      // new_dim
              result.data_,                               // result_data
              new_size                                  // total_elements
          );
          break;
        case 1:
          from_slice_kernel<1><<<blocks, threads_per_block>>>(
              a.data_,                                    // a_data
              d_a_strides,                          // a_strides
              d_starts,                              // starts
              d_new_strides,                     // new_strides
              d_new_shape,                      // new_dim
              result.data_,                               // result_data
              new_size                                  // total_elements
          );
          break;
        default:
            throw std::invalid_argument("Unsupported number of dimensions");
    }
    deallocate(d_a_strides);
    deallocate(d_starts);
    deallocate(d_new_strides);
    deallocate(d_new_shape);

    hipDeviceSynchronize();  // ⚠️ 强制同步
    // Step 5: Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err));
    }
    return result;
}

void TensorOpsCUDA::from_slice_backward(TensorImpl& ret, const TensorImpl& b,
                                              std::vector<int> starts, std::vector<int> ends) {
    int new_size = b.numel();
    int ndim = ret.shape().size();
    // Step 4: Launch kernel
    int threads_per_block = 256;
    int blocks = (new_size + threads_per_block - 1) / threads_per_block;
    int32_t *d_a_strides, *d_new_strides, *d_new_shape;
    int *d_starts;

    allocate(reinterpret_cast<void**>(&d_a_strides), ndim * sizeof(int32_t));
    allocate(reinterpret_cast<void**>(&d_starts), ndim * sizeof(int));
    allocate(reinterpret_cast<void**>(&d_new_strides), ndim * sizeof(int32_t));
    allocate(reinterpret_cast<void**>(&d_new_shape), ndim * sizeof(int32_t));

    copyHostToDevice(d_a_strides, ret.strides_.data(), ndim * sizeof(int32_t));
    copyHostToDevice(d_starts, starts.data(), ndim * sizeof(int));
    copyHostToDevice(d_new_strides, b.strides_.data(), ndim * sizeof(int32_t));
    copyHostToDevice(d_new_shape, b.shape().data(), ndim * sizeof(int32_t));

    switch (ndim) {
        case 4:
            from_slice_kernel_backward<4><<<blocks, threads_per_block>>>(
                ret.data_,                                    // a_data
                d_a_strides,                          // a_strides
                d_starts,                              // starts
                d_new_strides,                     // new_strides
                d_new_shape,                      // new_dim
                b.data_,                               // result_data
                new_size                                  // total_elements
            );
            break;
        case 5:
            from_slice_kernel_backward<5><<<blocks, threads_per_block>>>(
                ret.data_,                                    // a_data
                d_a_strides,                          // a_strides
                d_starts,                              // starts
                d_new_strides,                     // new_strides
                d_new_shape,                      // new_dim
                b.data_,                               // result_data
                new_size                                  // total_elements
            );
            break;
        case 2:
          from_slice_kernel_backward<2><<<blocks, threads_per_block>>>(
              ret.data_,                                    // a_data
              d_a_strides,                          // a_strides
              d_starts,                              // starts
              d_new_strides,                     // new_strides
              d_new_shape,                      // new_dim
              b.data_,                               // result_data
              new_size                                  // total_elements
          );
          break;
        case 1:
         from_slice_kernel_backward<1><<<blocks, threads_per_block>>>(
              ret.data_,                                    // a_data
              d_a_strides,                          // a_strides
              d_starts,                              // starts
              d_new_strides,                     // new_strides
              d_new_shape,                      // new_dim
              b.data_,                               // result_data
              new_size                                  // total_elements
          );
          break;
        default:
            throw std::invalid_argument("Unsupported number of dimensions");
    }
    deallocate(d_a_strides);
    deallocate(d_starts);
    deallocate(d_new_strides);
    deallocate(d_new_shape);

    hipDeviceSynchronize();  // ⚠️ 强制同步
    // Step 5: Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err));
    }

}

TensorImpl TensorOpsCUDA::concat(const TensorImpl& a , const TensorImpl& b, int32_t dim){
  Shape a_shape = a.shape();
  Shape b_shape = b.shape();
  Shape output_shape = a_shape;
  output_shape[dim] = a_shape[dim] + b_shape[dim];

  TensorImpl ret = TensorImpl::shape(output_shape, a.device(), a.type());
  using type = float;

  if (dim==a_shape.size()-1){
    size_t num_blocks = 1;
    size_t a_block_bytes = a_shape[dim] * sizeof(type);
    size_t b_block_bytes = b_shape[dim] * sizeof(type);

    for (int i = 0; i < a_shape.size() - 1; ++i) {
        num_blocks *= a_shape[i];
    }
    for (size_t i = 0; i < num_blocks; ++i) {
        const type* a_src = reinterpret_cast<type*>(a.data_) + i * a_shape[dim];
        const type* b_src = reinterpret_cast<type*>(b.data_) + i * b_shape[dim];
        type* output_dst = reinterpret_cast<type*>(ret.data_) + i * (a_shape[dim] + b_shape[dim]);
        hipMemcpyAsync(
            output_dst,
            a_src,
            a_block_bytes,
            hipMemcpyDeviceToDevice
        );
        hipMemcpyAsync(
            output_dst + a_shape[dim],
            b_src,
            b_block_bytes,
            hipMemcpyDeviceToDevice
        );
    }

     return ret;
  }
  if (dim == 1 && a_shape.size() == 4) {
     hipStream_t stream;
    hipStreamCreate(&stream);
    size_t num_samples = a_shape[0]; // N
    size_t a_sample_size = a.strides_[0] * sizeof(type);
    size_t b_sample_size = b.strides_[0] * sizeof(type);
    for (size_t i = 0; i < num_samples; ++i) {
        const type* a_src = reinterpret_cast<type*>(a.data_) + i * a.strides_[0];
        const type* b_src = reinterpret_cast<type*>(b.data_) + i * b.strides_[0];
        type* output_dst = reinterpret_cast<type*>(ret.data()) + i * (a.strides_[0] + b.strides_[0]);
        hipMemcpyAsync(output_dst, a_src, a_sample_size, hipMemcpyDeviceToDevice);
        hipMemcpyAsync(output_dst + a.strides_[0], b_src, b_sample_size, hipMemcpyDeviceToDevice);
    }
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    return ret;
  }
  if (dim == 2 && a_shape.size() == 4) {
    hipStream_t stream;
    hipStreamCreate(&stream);
    size_t N = a_shape[0];
    size_t C = a_shape[1];
    size_t H_a = a_shape[2];
    size_t W = a_shape[3];
    size_t H_b = b_shape[2];
    for (size_t i = 0; i < N; ++i) {
        for (size_t c = 0; c < C; ++c) {
            const type* a_src = reinterpret_cast<const type*>(a.data_) +
                i * a.strides_[0] + c * a.strides_[1];
            const type* b_src = reinterpret_cast<const type*>(b.data_) +
                i * b.strides_[0] + c * b.strides_[1];
            type* ret_dst = reinterpret_cast<type*>(ret.data()) +
                i * ret.strides_[0] + c * ret.strides_[1];
            hipMemcpyAsync(
                ret_dst,
                a_src,
                H_a * W * sizeof(type),
                hipMemcpyDeviceToDevice
            );
            hipMemcpyAsync(
                ret_dst + H_a * W,
                b_src,
                H_b * W * sizeof(type),
                hipMemcpyDeviceToDevice
            );
        }
    }

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    return ret;
    }
  else{
     throw std::invalid_argument("Unsupported dim, we only support last dim concat");
  }
}

std::vector<TensorImpl> TensorOpsCUDA::concat_backward(const TensorImpl& grad, int32_t dim, int32_t a_dim_shape){
  Shape grad_shape = grad.shape();
  Shape output_shape_1 = grad.shape();
  Shape output_shape_2 = grad.shape();
  int32_t b_dim_shape = output_shape_2[dim] - a_dim_shape;
  output_shape_1[dim] = a_dim_shape;
  output_shape_2[dim] = b_dim_shape;

  TensorImpl ret0 = TensorImpl::shape(output_shape_1, grad.device(), grad.type());
  TensorImpl ret1 = TensorImpl::shape(output_shape_2, grad.device(), grad.type());
  if (dim==grad_shape.size()-1){
    const int64_t num_dims = output_shape_1.size();
    int64_t inner_size = 1;
    for (int i = 0; i < num_dims - 1; ++i) {
        inner_size *= output_shape_1[i];
    }
    const int64_t a_dim_size = output_shape_1[dim];
    const int64_t b_dim_size = output_shape_2[dim];
    const int64_t concat_dim_size = a_dim_size + b_dim_size;
    for (int64_t i = 0; i < inner_size; ++i) {
        const float* grad_output_ptr = grad.data() + i * concat_dim_size;
        float* grad_a_ptr = ret0.data_ + i * a_dim_size;
        float* grad_b_ptr = ret1.data_ + i * b_dim_size;
        hipMemcpyAsync(
            grad_a_ptr,
            grad_output_ptr,
            a_dim_size * sizeof(float),
            hipMemcpyDeviceToDevice
        );
        hipMemcpyAsync(
            grad_b_ptr,
            grad_output_ptr + a_dim_size,
            b_dim_size * sizeof(float),
            hipMemcpyDeviceToDevice
        );

    }
    }
  else if (dim == 1 && grad_shape.size() == 4) {
   const int64_t total_elements = grad.numel();
    const int64_t N = grad_shape[0];
    const int64_t a_block_size = ret0.strides_[0];
    const int64_t b_block_size = ret1.strides_[0];
    const int64_t grad_block_size = ret0.strides_[0] + ret1.strides_[0];
    for (int64_t i = 0; i < N; ++i) {
      const float* grad_ptr = grad.data() + i * grad_block_size;
      float* grad_a_ptr = ret0.data_ + i * a_block_size;
      float* grad_b_ptr = ret1.data_ + i * b_block_size;
      hipMemcpyAsync(
          grad_a_ptr,
          grad_ptr,
          a_block_size * sizeof(float),
          hipMemcpyDeviceToDevice
      );
      hipMemcpyAsync(
          grad_b_ptr,
          grad_ptr + ret0.strides_[0],
          b_block_size * sizeof(float),
          hipMemcpyDeviceToDevice
      );
    }
  }
  else{
      throw std::invalid_argument("Unsupported dim, we only support last dim concat and dim == 1 in NCHW dim");
  }
  return {ret0, ret1};
}

TensorImpl TensorOpsCUDA::upsample_forward(const TensorImpl& a , int32_t scale_factor){
  TensorImpl ret = TensorImpl::shape({a.shape_[0], a.shape_[1], static_cast<int>(a.shape_[2]*scale_factor),
                                            static_cast<int>(a.shape_[3]*scale_factor)}, a.device());
  int32_t N = a.numel();
  int32_t h = a.shape_[2];
  int32_t w = a.shape_[3];
  if (scale_factor == 2 && N >= 256){
    dim3 grid(N / kBlockSize, 1);
    dim3 block(kBlockSize, 1);
    UpsampleNearest2D2XForward<<<grid, block>>>(N, a.data_, h,
                                                w, ret.data_);
    hipDeviceSynchronize();
  }
  else{}
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return ret;
}

TensorImpl TensorOpsCUDA::upsample_backward(const TensorImpl& a , int32_t scale_factor){
  TensorImpl ret = TensorImpl::shape({a.shape_[0], a.shape_[1], static_cast<int>(a.shape_[2]/scale_factor),
                                            static_cast<int>(a.shape_[3]/scale_factor)}, a.device());
  int32_t N = a.numel();
  int32_t h = ret.shape_[2];
  int32_t w = ret.shape_[3];
  if (scale_factor == 2 && N >= 256){
    dim3 grid(N / kBlockSize, 1);
    dim3 block(kBlockSize, 1);
    UpsampleNearest2D2XBackward<<<grid, block>>>(N, a.data_, h,
                                                 w, ret.data_);
  }
  else
      {}
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return ret;
}
TensorImpl  TensorOpsCUDA::flash_attention_(const TensorImpl& Q, const TensorImpl& K, const TensorImpl& V , int32_t head){
  throw std::runtime_error("We have not implement in CUDA yet");
}

const char* curandGetErrorString(hiprandStatus_t status) {
  switch (status) {
    case HIPRAND_STATUS_SUCCESS:
      return "HIPRAND_STATUS_SUCCESS";
    case HIPRAND_STATUS_VERSION_MISMATCH:
      return "HIPRAND_STATUS_VERSION_MISMATCH";
    case HIPRAND_STATUS_NOT_INITIALIZED:
      return "HIPRAND_STATUS_NOT_INITIALIZED";
    case HIPRAND_STATUS_ALLOCATION_FAILED:
      return "HIPRAND_STATUS_ALLOCATION_FAILED";
    case HIPRAND_STATUS_TYPE_ERROR:
      return "HIPRAND_STATUS_TYPE_ERROR";
    case HIPRAND_STATUS_OUT_OF_RANGE:
      return "HIPRAND_STATUS_OUT_OF_RANGE";
    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
      return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
      return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    case HIPRAND_STATUS_LAUNCH_FAILURE:
      return "HIPRAND_STATUS_LAUNCH_FAILURE";
    case HIPRAND_STATUS_PREEXISTING_FAILURE:
      return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case HIPRAND_STATUS_INITIALIZATION_FAILED:
      return "HIPRAND_STATUS_INITIALIZATION_FAILED";
    case HIPRAND_STATUS_ARCH_MISMATCH:
      return "HIPRAND_STATUS_ARCH_MISMATCH";
    case HIPRAND_STATUS_INTERNAL_ERROR:
      return "HIPRAND_STATUS_INTERNAL_ERROR";
  }
  return "Unknown cuRAND error";
}

const char* cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "Unknown cuBLAS error";
}

}  // namespace TinyTorch
