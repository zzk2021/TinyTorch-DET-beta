#include "hip/hip_runtime.h"
/*
 * TinyTorch
 * @author 	: keith@robot9.me
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#if CUDA_VERSION >= 12010
#include <hip/hip_fp8.h>
#endif

#include <cassert>
#include <cfloat>
#include <iostream>
#include "TensorImpl_cpu.h"
#include "TensorImpl_cuda.cuh"
#include "TensorImpl_cuda.inc"

namespace TinyTorch {

const char* curandGetErrorString(hiprandStatus_t status);
const char* cublasGetErrorString(hipblasStatus_t status);

#define DTYPE_CASE(dtype_enum, cuda_type, dtype)                              \
    case dtype_enum: dtype = cuda_type; break;                                \

//
#define DTYPE_SWITCH(dtype_var, dtype)                                        \
    switch (dtype_var) {                                                      \
        DTYPE_CASE(Dtype::float32,   HIP_R_32F, dtype)                       \
        DTYPE_CASE(Dtype::bfloat16,  HIP_R_16BF, dtype)                      \
        DTYPE_CASE(Dtype::float16,   HIP_R_16F, dtype)                       \
        default: throw std::invalid_argument("Unsupported Dtype");            \
    }                                                                         \

#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA error in file '" << __FILE__ << "' in line "         \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

#define CURAND_CHECK(call)                                               \
  do {                                                                   \
    hiprandStatus_t err = call;                                           \
    if (err != HIPRAND_STATUS_SUCCESS) {                                  \
      std::cerr << "CURAND error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << curandGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUBLAS_CHECK(call)                                               \
  do {                                                                   \
    hipblasStatus_t err = call;                                           \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                  \
      std::cerr << "CUBLAS error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << cublasGetErrorString(err) << " (" \
                << err << ")" << std::endl;                              \
      abort();                                                           \
    }                                                                    \
  } while (0)

#define CUDA_KERNEL_CHECK()                                                   \
  do {                                                                        \
    hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                 \
      std::cerr << "CUDA kernel error in file '" << __FILE__ << "' in line "  \
                << __LINE__ << ": " << hipGetErrorString(err) << " (" << err \
                << ")" << std::endl;                                          \
      abort();                                                                \
    }                                                                         \
  } while (0)

static std::random_device _r;
unsigned long RandomGeneratorCUDA::seed_ = _r();
unsigned long RandomGeneratorCUDA::sequence_ = 0;

void* AllocatorCPU::allocatePinned(size_t size) {
  void* ptr = nullptr;
  CUDA_CHECK(hipHostMalloc(&ptr, size));
  return ptr;
}

void AllocatorCPU::deallocatePinned(void* ptr) {
  CUDA_CHECK(hipHostFree(ptr));
}

void AllocatorCUDA::allocate(void** ptr, size_t size) {
  CUDA_CHECK(hipMalloc(ptr, size));
}

void AllocatorCUDA::deallocate(void* ptr) {
  if (ptr) {
    CUDA_CHECK(hipFree(ptr));
  }
}

TensorOpsCUDA::TensorOpsCUDA(int32_t device, size_t blockSize)
    : cudaDeviceIdx_(device), blockSize_(blockSize) {
  CUDA_CHECK(hipSetDevice(cudaDeviceIdx_));
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp_, cudaDeviceIdx_));

  if (blockSize_ > deviceProp_.maxThreadsPerBlock) {
    blockSize_ = deviceProp_.maxThreadsPerBlock;
  }

  allocator_.setBaseAllocator(std::make_shared<AllocatorCUDA>());
}

TensorOpsCUDA::~TensorOpsCUDA() {
  allocator_.clear();
  if (blasHandle_) {
    hipblasDestroy(blasHandle_);
  }
}

hipblasHandle_t TensorOpsCUDA::getCublasHandle() {
  if (blasHandle_ == nullptr) {
    hipblasCreate(&blasHandle_);
  }
  return blasHandle_;
}

template <typename T>
TensorCudaCtx<T> TensorOpsCUDA::getTensorCtx(const TensorImpl& t) {
  TensorCudaCtx<T> ret{};
  ret.dimCount_ = t.dimCount_;
  ret.elemCount_ = t.elemCount_;
  memcpy(ret.shape_, t.shape_.data(), t.dimCount_ * sizeof(int32_t));
  memcpy(ret.strides_, t.strides_.data(), t.dimCount_ * sizeof(int32_t));
  ret.data_ = reinterpret_cast<T*>(t.data_);
  return ret;
}

template <typename OP>
void TensorOpsCUDA::opSingle_(TensorImpl& t) const {
  if (t.type_ == Dtype::float16)
    kSingleOp_<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
       reinterpret_cast<half*>(t.data_), t.elemCount_);
  if (t.type_ == Dtype::bfloat16)
    kSingleOp_<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*> (t.data_) , t.elemCount_);
  else
    kSingleOp_<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
       t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opSingle(const TensorImpl& t) const {
  auto result = TensorImpl::shape(t.shape(), t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    kSingleOp<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(t.data_), t.elemCount_);
  if (t.type_ == Dtype::bfloat16)
    kSingleOp<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_),reinterpret_cast<__hip_bfloat16*> (t.data_) , t.elemCount_);
  else
    kSingleOp<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      result.data_, t.data_, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a,
                                 const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_, a.type_);
  if (a.type() == Dtype::float32)
    kPairOp<OP, float><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  else if (a.type() == Dtype::bfloat16){
    kPairOp<OP, __hip_bfloat16><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      reinterpret_cast<__hip_bfloat16*>(b.data_), result.elemCount_);
  }
  else if (a.type() == Dtype::float16){
    kPairOp<OP, half><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      reinterpret_cast<half*>(b.data_), result.elemCount_);
  }
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(const TensorImpl& a, float b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_, a.type_);

  if (a.type() == Dtype::float32)
    kPairScalarSecondOp<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b, a.elemCount_);

  else if (a.type() == Dtype::bfloat16)
    kPairScalarSecondOp<OP, __hip_bfloat16><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      static_cast<__hip_bfloat16>(b), a.elemCount_);

  else if (a.type() == Dtype::float16)
    kPairScalarSecondOp<OP, half><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      static_cast<half>(b), a.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPair(float a, const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_, b.type_);

  if (b.type() == Dtype::float32)
    kPairScalarFirstOp<OP, float><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      result.data_, a, b.data_, b.elemCount_);

  else if (b.type() == Dtype::bfloat16)
    kPairScalarFirstOp<OP, __hip_bfloat16><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_),
       static_cast<__hip_bfloat16>(a),
      reinterpret_cast<__hip_bfloat16*>(b.data_), b.elemCount_);

  else if (b.type() == Dtype::float16)
    kPairScalarFirstOp<OP, half><<<getGridSize(b.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_),
      static_cast<half>(a),
      reinterpret_cast<half*>(b.data_), b.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarFirst(const TensorImpl& a,
                                            const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape(), b.device_, b.type_);

  if (b.type() == Dtype::float32)
    kPairScalarFirstOp<OP, float><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);

  else if (b.type() == Dtype::bfloat16)
    kPairScalarFirstOp<OP, __hip_bfloat16><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(result.data_), reinterpret_cast<__hip_bfloat16*>(a.data_),
      reinterpret_cast<__hip_bfloat16*>(b.data_), result.elemCount_);

  else if (b.type() == Dtype::float16)
    kPairScalarFirstOp<OP, half><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(result.data_), reinterpret_cast<half*>(a.data_),
      reinterpret_cast<half*>(b.data_), result.elemCount_);

  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairScalarSecond(const TensorImpl& a,
                                             const TensorImpl& b) const {
  auto result = TensorImpl::shape(a.shape(), a.device_);
  kPairScalarSecondOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  return result;
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, float b) const {

  if (t.type() == Dtype::float32)
    kPairScalarSecondOp_<OP, float><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      t.data_, b, t.elemCount_);

  else if (t.type() == Dtype::bfloat16)
    kPairScalarSecondOp_<OP, __hip_bfloat16><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(t.data_), __float2bfloat16(b),
      t.elemCount_);

  else if (t.type() == Dtype::float16)
    kPairScalarSecondOp_<OP, half><<<getGridSize(t.elemCount_), getBlockSize()>>>(
      reinterpret_cast<half*>(t.data_), __float2half(b),
      t.elemCount_);

  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPair_(TensorImpl& t, const TensorImpl& b) const {
  kPairOp_<OP><<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, b.data_,
                                                              t.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
void TensorOpsCUDA::opPairScalarFirst_(TensorImpl& a,
                                       const TensorImpl& b) const {
  auto result = TensorImpl::shape(b.shape_, b.device_);
  kPairScalarFirstOp<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      result.data_, a.data_, b.data_, result.elemCount_);
  CUDA_KERNEL_CHECK();
  a = std::move(result);
}

template <typename OP>
void TensorOpsCUDA::opPairScalarSecond_(TensorImpl& a,
                                        const TensorImpl& b) const {
  kPairScalarSecondOp_<OP><<<getGridSize(a.elemCount_), getBlockSize()>>>(
      a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP, typename T>
void TensorOpsCUDA::broadcastImpl(TensorImpl& result, const TensorImpl& a,
                                  const TensorImpl& b) const {
  // fast broadcast with a

  if (b.elemCount_ == result.elemCount_) {

    if (isLeadingOnes(a.shape())) {
        kBroadcastOpFast<OP, true, true, T>
              <<<getGridSize(result.elemCount_), getBlockSize()>>>(
                  reinterpret_cast<T*>(result.data_),
                  reinterpret_cast<T*>(a.data_),
                  reinterpret_cast<T*>(b.data_),
                  a.elemCount_, result.elemCount_);

      CUDA_KERNEL_CHECK();
      return;
    }
    if (isTrailingOnes(a.shape())) {
      kBroadcastOpFast<OP, false, true, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              result.elemCount_ / a.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  // fast broadcast with b
  if (a.elemCount_ == result.elemCount_) {
    if (isLeadingOnes(b.shape())) {
      kBroadcastOpFast<OP, true, false, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              b.elemCount_, result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }

    if (isTrailingOnes(b.shape())) {
      kBroadcastOpFast<OP, false, false, T>
          <<<getGridSize(result.elemCount_), getBlockSize()>>>(
              reinterpret_cast<T*>(result.data_),
              reinterpret_cast<T*>(a.data_),
              reinterpret_cast<T*>(b.data_),
              result.elemCount_ / b.elemCount_,
              result.elemCount_);
      CUDA_KERNEL_CHECK();
      return;
    }
  }

  const auto ctxA = getTensorCtx<T>(a);
  const auto ctxB = getTensorCtx<T>(b);
  const auto ctxC = getTensorCtx<T>(result);
  kBroadcastOpCommon<OP><<<getGridSize(result.elemCount_), getBlockSize()>>>(
      ctxC, ctxA, ctxB, result.elemCount_);
  CUDA_KERNEL_CHECK();
}

template <typename OP>
TensorImpl TensorOpsCUDA::opPairBroadcast(const TensorImpl& a,
                                          const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return {};
  }

  if (comp == ShapeCompatible_SameShape) {
    return opPair<OP>(a, b);
  }

  auto result = TensorImpl::shape(retShape, a.device_);
  broadcastImpl<OP>(result, a, b);
  return result;
}



template <typename OP>
void TensorOpsCUDA::opPairBroadcast_(TensorImpl& a, const TensorImpl& b) const {
  Shape retShape;
  auto comp = checkShapeCompatible(a.shape(), b.shape(), retShape);
  if (comp == ShapeCompatible_Error) {
    error(__FUNCTION__, TensorError_ShapeNotAligned);
    return;
  }
  if (comp == ShapeCompatible_SameShape) {
    opPair_<OP>(a, b);
    return;
  }
  auto result = TensorImpl::shape(retShape, a.device_, a.type_);

  if (a.type_ == Dtype::float32)
    broadcastImpl<OP>(result, a, b);
  else if (a.type_ == Dtype::float16)
    broadcastImpl<OP, half>(result, a, b);
  else if (a.type_ == Dtype::bfloat16)
    broadcastImpl<OP, __hip_bfloat16>(result, a, b);

  a = std::move(result);
}

template <typename OP ,typename T>
void TensorOpsCUDA::reduceAllImpl(T* dOutput, const T* dInput,
                                  int32_t n, int32_t m, KernelFunc<OP, T> kernel) {
  auto blocks = getGridSize(n);

  T* dTmp = nullptr;
  allocate(reinterpret_cast<void**>(&dTmp), m * blocks * sizeof(T));

  kernel<<<m * blocks, getBlockSize()>>>(dTmp, dInput, n, m);
  CUDA_KERNEL_CHECK();
  while (blocks > 1) {
    auto currBlocks = blocks;
    blocks = getGridSize(currBlocks);
    kReduceAll<OP><<<m * blocks, getBlockSize()>>>(dTmp, dTmp, currBlocks, m);
    CUDA_KERNEL_CHECK();
  }
  copyOnDevice(dOutput, dTmp, m * sizeof(T));
  deallocate(dTmp);
}

template <typename OP, typename T>
void TensorOpsCUDA::reduceAll(T* dOutput, const T* dInput, int32_t n,
                              int32_t m) {
  reduceAllImpl<OP, T>(dOutput, dInput, n, m, kReduceAll<OP, T>);
}

template <typename OP>
void TensorOpsCUDA::reduceAllIdx(float* dOutput, const float* dInput, int32_t n,
                                 int32_t m) {
  reduceAllImpl<OP>(dOutput, dInput, n, m, kReduceAllIdx<OP>);
}

template <typename OP, typename T>
void TensorOpsCUDA::reduceAllLastDim(T* dOutput, const T* dInput,
                                     int32_t n, int32_t m) {
  reduceAllImpl<OP, T>(dOutput, dInput, n, m, kReduceAllLastDim<OP, T>);

}

template <typename OP, typename T>
std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::reduceDim(const TensorImpl& t,
                                                           int32_t dim,
                                                           bool keepDims) {
  if (dim < 0) {
    dim += t.dimCount_;
  }
  if (dim < 0 || dim >= t.dimCount_) {
    error(__FUNCTION__, TensorError_InvalidAxis);
    return {};
  }

  const auto retShape = getReduceShape(t, dim, false);
  auto values = TensorImpl::shape(retShape, t.device_, t.type_);
  auto indices = TensorImpl::shape(retShape, t.device_);

  if (dim == t.dimCount_ - 1) {
    kReduceLastDim<OP, T><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        reinterpret_cast<T*>(values.data_), indices.data_, reinterpret_cast<T*>(t.data_),
        t.shape_[dim], values.elemCount_);
  } else {
    auto ctxT = getTensorCtx<T>(t);
    auto ctxValues = getTensorCtx<T>(values);
    kReduceDim<OP, T><<<getGridSize(t.elemCount_), getBlockSize()>>>(
        ctxValues, indices.data_, ctxT, dim, ctxValues.elemCount_);
  }
  CUDA_KERNEL_CHECK();

  if (keepDims) {
    const auto shapeKeepDims = getReduceShape(t, dim, true);
    values.reshape_(shapeKeepDims);
    indices.reshape_(shapeKeepDims);
  }

  // values as the same type as origin data
  // but indices is float32
  return {values, indices};
}

template <typename T>
void TensorOpsCUDA::transpose2D(T* out, const T* in, int32_t width,
                                int32_t height) {
  dim3 blockSize(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_DIM);
  dim3 gridSize((width + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM,
                (height + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM);
  kTranspose<<<gridSize, blockSize>>>(out, in, width, height);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::allocate(void** ptr, size_t size) {
  allocator_.allocate(ptr, size);
}

void TensorOpsCUDA::deallocate(void* ptr) { allocator_.deallocate(ptr); }

void TensorOpsCUDA::copyHostToDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
}

void TensorOpsCUDA::convertTypeOnDevice(void* dst, void* src, size_t count, Dtype Ti ,Dtype To) {
   // src = static_cast<float*>(src);
    size_t threads_per_block = 256;
    size_t blocks = (count + threads_per_block - 1) / threads_per_block;
    if (Ti == Dtype::float32) {
        assert(static_cast<float*>(src) != nullptr);
    } else if (Ti == Dtype::bfloat16) {
        assert(static_cast<__hip_bfloat16*>(src) != nullptr);
    } else if (Ti == Dtype::float16) {
        assert(static_cast<half*>(src) != nullptr);
    }

    if (To == Dtype::float32) {
        assert(static_cast<float*>(dst) != nullptr);
    } else if (To == Dtype::bfloat16) {
        assert(static_cast<__hip_bfloat16*>(dst) != nullptr);
    } else if (To == Dtype::float16) {
        assert(static_cast<half*>(dst) != nullptr);
    }

    if (Ti == Dtype::float32 && To == Dtype::float16) {
        convertFloatToHalfKernel<<<blocks, threads_per_block>>>(
            static_cast<float*>(src), static_cast<half*>(dst), count);
    } else if (Ti == Dtype::float32 && To == Dtype::bfloat16) {
        convertFloatToBf16Kernel<<<blocks, threads_per_block>>>(
            static_cast<float*>(src), static_cast<__hip_bfloat16*>(dst), count);
    } else if (Ti == Dtype::bfloat16 && To == Dtype::float32) {
        convertBf16ToFloatKernel<<<blocks, threads_per_block>>>(
            static_cast<__hip_bfloat16*>(src), static_cast<float*>(dst), count);
    } else if (Ti == Dtype::float16 && To == Dtype::float32) {
        convertHalfToFloatKernel<<<blocks, threads_per_block>>>(
            static_cast<half*>(src), static_cast<float*>(dst), count);
    } else if (Ti == To) {
        if (Ti == Dtype::float32)
            hipMemcpy(dst, src, count * sizeof(float), hipMemcpyDeviceToDevice);
        else if (Ti == Dtype::bfloat16 || Ti == Dtype::float16)
            hipMemcpy(dst, src, count * sizeof(half), hipMemcpyDeviceToDevice);
    } else {
        LOGW("Type conversion from %d to %d is not supported, keeping the same type",
             Ti, To);
        //
    }
    CUDA_KERNEL_CHECK();
}



void TensorOpsCUDA::copyOnDevice(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice));
}

void TensorOpsCUDA::copyDeviceToHost(void* dst, const void* src, size_t count) {
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
}

void TensorOpsCUDA::fillConstant_(float* dst, float val, size_t count) {
  kFillConstant<<<getGridSize(count, 4), getBlockSize()>>>(dst, val, count);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillConstant_(TensorImpl& t, float val) {
  kFillConstant<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(t.data_, val,
                                                                  t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillLinSpace_(float* dst, float start, float step,
                                  size_t count) {
  kFillLinSpace<<<getGridSize(count, 4), getBlockSize()>>>(dst, start, step,
                                                           count);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandUniform_(TensorImpl& t, float min, float max) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandUniform<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, min, max, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandNormal_(TensorImpl& t) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandNormal<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, 0.f, 1.f, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::fillRandBernoulli_(TensorImpl& t, float p) {
  auto seed = RandomGeneratorCUDA::getSeed();
  auto seq = RandomGeneratorCUDA::nextSequence();
  kFillRandBernoulli<<<getGridSize(t.elemCount_, 4), getBlockSize()>>>(
      t.data_, p, seed, seq, t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaAdd>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaAdd>(a, b);
  }
  return opPairBroadcast<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaSub>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaSub>(a, b);
  }
  return opPairBroadcast<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaMul>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMul>(a, b);
  }
  return opPairBroadcast<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    return opPairScalarFirst<OpCudaDiv>(a, b);
  }
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaDiv>(a, b);
  }
  return opPairBroadcast<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaPow>(a, b);
  }
  return opPairBroadcast<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const TensorImpl& a, const float& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const TensorImpl& a, const float& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const TensorImpl& a, const float& b) {
  return opPair<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::pow(const TensorImpl& a, const float& b) {
  return opPair<OpCudaPow>(a, b);
}

TensorImpl TensorOpsCUDA::add(const float& a, const TensorImpl& b) {
  return opPair<OpCudaAdd>(a, b);
}

TensorImpl TensorOpsCUDA::sub(const float& a, const TensorImpl& b) {
  return opPair<OpCudaSub>(a, b);
}

TensorImpl TensorOpsCUDA::mul(const float& a, const TensorImpl& b) {
  return opPair<OpCudaMul>(a, b);
}

TensorImpl TensorOpsCUDA::div(const float& a, const TensorImpl& b) {
  return opPair<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaAdd>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaAdd>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaSub>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaSub>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaMul>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaMul>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const TensorImpl& b) {
  if (a.dimCount_ == 0) {
    opPairScalarFirst_<OpCudaDiv>(a, b);
    return;
  }
  if (b.dimCount_ == 0) {
    opPairScalarSecond_<OpCudaDiv>(a, b);
    return;
  }
  opPairBroadcast_<OpCudaDiv>(a, b);
}

void TensorOpsCUDA::add_(TensorImpl& a, const float& b) {
  opPair_<OpCudaAdd>(a, b);
}

void TensorOpsCUDA::sub_(TensorImpl& a, const float& b) {
  opPair_<OpCudaSub>(a, b);
}

void TensorOpsCUDA::mul_(TensorImpl& a, const float& b) {
  opPair_<OpCudaMul>(a, b);
}

void TensorOpsCUDA::div_(TensorImpl& a, const float& b) {
  opPair_<OpCudaDiv>(a, b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaEq>(a, b);
  }
  return opPairBroadcast<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaNe>(a, b);
  }
  return opPairBroadcast<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGe>(a, b);
  }
  return opPairBroadcast<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaGt>(a, b);
  }
  return opPairBroadcast<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLe>(a, b);
  }
  return opPairBroadcast<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaLt>(a, b);
  }
  return opPairBroadcast<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMax>(a, b);
  }
  return opPairBroadcast<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const TensorImpl& b) {
  if (b.dimCount_ == 0) {
    return opPairScalarSecond<OpCudaMin>(a, b);
  }
  return opPairBroadcast<OpCudaMin>(a, b);
}

TensorImpl TensorOpsCUDA::eq(const TensorImpl& a, const float& b) {
  return opPair<OpCudaEq>(a, b);
}

TensorImpl TensorOpsCUDA::ne(const TensorImpl& a, const float& b) {
  return opPair<OpCudaNe>(a, b);
}

TensorImpl TensorOpsCUDA::ge(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGe>(a, b);
}

TensorImpl TensorOpsCUDA::gt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaGt>(a, b);
}

TensorImpl TensorOpsCUDA::le(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLe>(a, b);
}

TensorImpl TensorOpsCUDA::lt(const TensorImpl& a, const float& b) {
  return opPair<OpCudaLt>(a, b);
}

TensorImpl TensorOpsCUDA::maximum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMax>(a, b);
}

TensorImpl TensorOpsCUDA::minimum(const TensorImpl& a, const float& b) {
  return opPair<OpCudaMin>(a, b);
}

void TensorOpsCUDA::sin_(TensorImpl& t) { opSingle_<OpCudaSin_>(t); }

void TensorOpsCUDA::cos_(TensorImpl& t) { opSingle_<OpCudaCos_>(t); }

void TensorOpsCUDA::sqrt_(TensorImpl& t) { opSingle_<OpCudaSqrt_>(t); }

void TensorOpsCUDA::tanh_(TensorImpl& t) { opSingle_<OpCudaTanh_>(t); }

void TensorOpsCUDA::exp_(TensorImpl& t) { opSingle_<OpCudaExp_>(t); }

void TensorOpsCUDA::log_(TensorImpl& t) { opSingle_<OpCudaLog_>(t); }

TensorImpl TensorOpsCUDA::sin(const TensorImpl& t) {
  return opSingle<OpCudaSin>(t);
}

TensorImpl TensorOpsCUDA::cos(const TensorImpl& t) {
  return opSingle<OpCudaCos>(t);
}

TensorImpl TensorOpsCUDA::sqrt(const TensorImpl& t) {
  return opSingle<OpCudaSqrt>(t);
}

TensorImpl TensorOpsCUDA::tanh(const TensorImpl& t) {
  return opSingle<OpCudaTanh>(t);
}

TensorImpl TensorOpsCUDA::exp(const TensorImpl& t) {
  return opSingle<OpCudaExp>(t);
}

TensorImpl TensorOpsCUDA::log(const TensorImpl& t) {
  return opSingle<OpCudaLog>(t);
}

void TensorOpsCUDA::clampMin_(TensorImpl& t, float min) {
  opPair_<OpCudaMax>(t, min);
}

void TensorOpsCUDA::clampMax_(TensorImpl& t, float max) {
  opPair_<OpCudaMin>(t, max);
}

void TensorOpsCUDA::clamp_(TensorImpl& t, float min, float max) {
  kClamp_<<<getGridSize(t.elemCount_), getBlockSize()>>>(t.data_, min, max,
                                                         t.elemCount_);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::clampMin(const TensorImpl& t, float min) {
  return opPair<OpCudaMax>(t, min);
}

TensorImpl TensorOpsCUDA::clampMax(const TensorImpl& t, float max) {
  return opPair<OpCudaMin>(t, max);
}

TensorImpl TensorOpsCUDA::clamp(const TensorImpl& t, float min, float max) {
  auto ret = TensorImpl::shape(t.shape_, t.device_);
  kClamp<<<getGridSize(t.elemCount_), getBlockSize()>>>(ret.data_, t.data_, min,
                                                        max, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::min(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAll<OpCudaReduceMin>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::max(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceMax>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceMax>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceMax>(ret.data_, t.data_, t.elemCount_);
  return ret;
}


TensorImpl TensorOpsCUDA::sum(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceSum>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return t;
  }
  auto ret = TensorImpl::scalar(t.device_, t.type_);

  if (t.type_ == Dtype::float16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<half*>(ret.data_),reinterpret_cast<half*>(t.data_), t.elemCount_);
  else if (t.type_ == Dtype::bfloat16)
    reduceAll<OpCudaReduceSum>(reinterpret_cast<__hip_bfloat16*>(ret.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), t.elemCount_);
  else
    reduceAll<OpCudaReduceSum>(ret.data_, t.data_, t.elemCount_);

  const auto r = 1.f / static_cast<float>(t.elemCount_);
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::var(const TensorImpl& t, bool unbiased) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  const auto meanVal = mean(t);
  const auto squaredDiff = TensorImpl::shape({t.elemCount_}, t.device_);
  kSquaredDiff<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      squaredDiff.data_, t.data_, meanVal.data_, t.elemCount_);

  auto ret = TensorImpl::scalar(t.device_);
  reduceAll<OpCudaReduceSum>(ret.data_, squaredDiff.data_, t.elemCount_);

  const auto n = static_cast<float>(t.elemCount_);
  auto r = 1.f / n;
  if (unbiased) {
    r *= n / (n - 1.f);
  }
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::argmin(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAllIdx<OpCudaReduceMin>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

TensorImpl TensorOpsCUDA::argmax(const TensorImpl& t) {
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }
  auto ret = TensorImpl::scalar(t.device_);
  reduceAllIdx<OpCudaReduceMax>(ret.data_, t.data_, t.elemCount_);
  return ret;
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::min(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_)};
  }
  return reduceDim<OpCudaReduceMin>(t, dim, keepDims);
}



std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::max(const TensorImpl& t,
                                                     int32_t dim,
                                                     bool keepDims) {
  if (t.dimCount_ == 0) {
    return {t, TensorImpl::scalar(0, t.device_, t.type_)};
  }
  if (t.type_ == Dtype::float16)
    return reduceDim<OpCudaReduceMax, half>(t, dim, keepDims);
  else if(t.type_ == Dtype::bfloat16)
    return reduceDim<OpCudaReduceMax, __hip_bfloat16>(t, dim, keepDims);
  else
    return reduceDim<OpCudaReduceMax>(t, dim, keepDims);
}

TensorImpl TensorOpsCUDA::sum(const TensorImpl& t,
                              const std::vector<int32_t>& dims, bool keepDims) {
  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }
  if (t.dimCount_ == 0) {
    return t;
  }

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto ret = TensorImpl::shape(retShape, t.device_);

  if (dims.size() == 1) {
    auto d = dims[0];
    if (d < 0) {
      d += t.dimCount_;
    }

    // first dim
    if (d == 0) {
      const auto dimSize = t.shape_.front();
      const auto tmp = TensorImpl::shape(t.shape_, t.device_, t.type_);
      if (t.type_ == Dtype::float16)
        transpose2D(reinterpret_cast<half*>(tmp.data_),
                    reinterpret_cast<half*>(t.data_),
                    ret.elemCount_,
                    dimSize);
      else if (t.type_ == Dtype::bfloat16)
        transpose2D(reinterpret_cast<__hip_bfloat16*>(tmp.data_),
                    reinterpret_cast<__hip_bfloat16*>(t.data_),
                    ret.elemCount_,
                    dimSize);
      else
        transpose2D(tmp.data_, t.data_, ret.elemCount_, dimSize);


      if (t.type_ == Dtype::float16)
        reduceAllLastDim<OpCudaReduceSum, half>(reinterpret_cast<half*>(ret.data_),
                    reinterpret_cast<half*>(tmp.data_),
                    dimSize,
                    ret.elemCount_);
      else if (t.type_ == Dtype::bfloat16)
        reduceAllLastDim<OpCudaReduceSum, __hip_bfloat16>(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                    reinterpret_cast<__hip_bfloat16*>(tmp.data_),
                    dimSize,
                    ret.elemCount_);
      else
        reduceAllLastDim<OpCudaReduceSum, float>(ret.data_,
                    tmp.data_,
                    dimSize,
                    ret.elemCount_);

      return ret;
    }

    // last dim
    if (d == t.dimCount_ - 1) {
      const auto dimSize = t.shape_.back();
      if (t.type_ == Dtype::float16)
        reduceAllLastDim<OpCudaReduceSum, half>(reinterpret_cast<half*>(ret.data_),
                    reinterpret_cast<half*>(t.data_),
                    dimSize,
                    ret.elemCount_);
      else if (t.type_ == Dtype::bfloat16)
        reduceAllLastDim<OpCudaReduceSum, __hip_bfloat16>(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                    reinterpret_cast<__hip_bfloat16*>(t.data_),
                    dimSize,
                    ret.elemCount_);
      else
        reduceAllLastDim<OpCudaReduceSum, float>(ret.data_,
                    t.data_,
                    dimSize,
                    ret.elemCount_);
      return ret;
    }
  }

  auto ctxT = getTensorCtx(t);
  fillConstant_(ret, 0);
  kReduceSum<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ret.data_, ctxT, inAxis, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::mean(const TensorImpl& t,
                               const std::vector<int32_t>& dims,
                               bool keepDims) {
  auto ret = sum(t, dims, keepDims);
  if (!ret.empty()) {
    auto reduceSize = (float)t.elemCount_ / (float)ret.elemCount_;
    auto r = 1.f / reduceSize;
    mul_(ret, r);
  }
  return ret;
}

TensorImpl TensorOpsCUDA::var(const TensorImpl& t,
                              const std::vector<int32_t>& dims, bool unbiased,
                              bool keepDims) {
  FixedVector<uint8_t> inAxis{};
  for (int32_t d : dims) {
    if (d < 0) {
      d += t.dimCount_;
    }
    if (d < 0 || d >= t.dimCount_) {
      error(__FUNCTION__, TensorError_InvalidAxis);
      return {};
    }
    inAxis.data[d] = 1;
  }
  if (t.dimCount_ == 0) {
    return TensorImpl::scalar(0, t.device_);
  }

  auto retShape = getReduceShape(t, inAxis, keepDims);
  auto ret = TensorImpl::shape(retShape, t.device_);

  auto meanTensor = mean(t, dims, true);
  fillConstant_(ret, 0);

  auto ctxT = getTensorCtx(t);
  kReduceVar<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ret.data_, ctxT, meanTensor.data_, inAxis, t.elemCount_);
  CUDA_KERNEL_CHECK();

  auto reduceSize = (float)t.elemCount_ / (float)ret.elemCount_;
  auto r = 1.f / reduceSize;
  if (unbiased) {
    r *= reduceSize / (reduceSize - 1.f);
  }
  mul_(ret, r);
  return ret;
}

TensorImpl TensorOpsCUDA::permute(const TensorImpl& t,
                                  const std::vector<int32_t>& dims) {
  auto retShape = t.shape_;
  reorderIndices(retShape.data(), dims);
  auto ret = TensorImpl::shape(retShape, t.device_, t.type_);

  auto ctxT = getTensorCtx(t);
  auto ctxRet = getTensorCtx(ret);

  auto* dimsDataPtr = (FixedVector<int32_t>*)dims.data();
  kPermute<<<getGridSize(t.elemCount_), getBlockSize()>>>(
      ctxRet, ctxT, *dimsDataPtr, t.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}


TensorImpl TensorOpsCUDA::transpose2D(const TensorImpl& t) {
  auto ret = TensorImpl::shape({t.shape_[1], t.shape_[0]}, t.device_, t.type_);
  if (t.type_ == Dtype::float16)
      transpose2D(reinterpret_cast<half*>(ret.data_),
                  reinterpret_cast<half*>(t.data_), t.shape_[1], t.shape_[0]);
  else if (t.type_ == Dtype::bfloat16)
      transpose2D(reinterpret_cast<__hip_bfloat16*>(ret.data_),
                  reinterpret_cast<__hip_bfloat16*>(t.data_), t.shape_[1], t.shape_[0]);
  else
      transpose2D(ret.data_, t.data_, t.shape_[1], t.shape_[0]);
  return ret;
}

TensorImpl TensorOpsCUDA::index(
    const TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  Shape retShape = {fistDim};
  for (auto i = len; i < t.dimCount_; i++) {
    retShape.push_back(t.shape_[i]);
  }
  auto retTensor = TensorImpl::shape(retShape, t.device_);

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndex<<<getGridSize(fistDim), getBlockSize()>>>(
      retTensor.data_, ctxT, indicesData, dimStride, len, fistDim);
  CUDA_KERNEL_CHECK();
  return retTensor;
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices, float val) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(fistDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val, fistDim);
  CUDA_KERNEL_CHECK();
}

void TensorOpsCUDA::indexPut_(
    TensorImpl& t,
    const std::vector<std::reference_wrapper<TensorImpl>>& indices,
    const TensorImpl& val) {
  auto len = (int32_t)indices.size();
  auto fistDim = (int32_t)indices[0].get().elemCount_;
  auto dimStride = t.strides_[len - 1];
  assert(val.elemCount_ == dimStride * fistDim);

  FixedVector<float*> indicesData{};
  for (int32_t i = 0; i < len; i++) {
    indicesData.data[i] = indices[i].get().data_;
  }
  auto ctxT = getTensorCtx(t);
  kIndexPut<<<getGridSize(fistDim), getBlockSize()>>>(
      ctxT, indicesData, dimStride, len, val.data_, fistDim);
  CUDA_KERNEL_CHECK();
}

TensorImpl TensorOpsCUDA::im2col(const TensorImpl& t, Size2D kernel,
                                 Size2D stride, Size2D padding) {
  // this: [C, H, W], [N, C, H, W]
  assert(t.dimCount_ == 3 || t.dimCount_ == 4);
  int32_t batch = (t.dimCount_ == 4) ? t.shape_[0] : 1;
  int32_t channels = (t.dimCount_ == 4) ? t.shape_[1] : t.shape_[0];
  int32_t height = (t.dimCount_ == 4) ? t.shape_[2] : t.shape_[1];
  int32_t width = (t.dimCount_ == 4) ? t.shape_[3] : t.shape_[2];
  int32_t outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  int32_t outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  int32_t colH = outH * outW;
  int32_t colW = channels * kernel.h * kernel.w;
  auto retTensor = TensorImpl::shape({batch * colH, colW}, t.device_);

  int32_t imStride = t.strides_[0];
  int totalElements = batch * outH * outW * channels * kernel.h * kernel.w;

  if (t.type_ == Dtype::float32)
    kIm2Col<<<getGridSize(totalElements), getBlockSize()>>>(
      retTensor.data_, t.data_, batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
  else if (t.type_ == Dtype::float16){
      retTensor.to_(Dtype::float16);
    kIm2Col<half><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<half*>(retTensor.data_), reinterpret_cast<half*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
    }
  else if (t.type_ == Dtype::bfloat16){
      retTensor.to_(Dtype::bfloat16);
    kIm2Col<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(retTensor.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colH, colW);
    }

  CUDA_KERNEL_CHECK();
  return retTensor;
}

TensorImpl TensorOpsCUDA::col2im(const TensorImpl& t, const Shape& shape,
                                 Size2D kernel, Size2D stride, Size2D padding) {
  // shape: [C, H, W], [N, C, H, W]
  assert(shape.size() == 3 || shape.size() == 4);
  int32_t batch = (shape.size() == 4) ? shape[0] : 1;
  int32_t channels = (shape.size() == 4) ? shape[1] : shape[0];
  int32_t height = (shape.size() == 4) ? shape[2] : shape[1];
  int32_t width = (shape.size() == 4) ? shape[3] : shape[2];

  auto outH = (height - kernel.h + 2 * padding.h) / stride.h + 1;
  auto outW = (width - kernel.w + 2 * padding.w) / stride.w + 1;

  // int32_t colH = outH * outW;
  int32_t colW = channels * kernel.h * kernel.w;

  auto retTensor = TensorImpl::zeros(shape, t.device_);

  auto imStride = retTensor.strides_[0];
  int totalElements = batch * channels * height * width;

  if (t.type_ == Dtype::float32)
    kCol2Im<<<getGridSize(totalElements), getBlockSize()>>>(
      retTensor.data_, t.data_, batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
  else if (t.type_ == Dtype::float16){
      retTensor.to_(Dtype::float16);
    kCol2Im<half><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<half*>(retTensor.data_), reinterpret_cast<half*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
    }
  else if (t.type_ == Dtype::bfloat16){
      retTensor.to_(Dtype::bfloat16);
    kCol2Im<__hip_bfloat16><<<getGridSize(totalElements), getBlockSize()>>>(
      reinterpret_cast<__hip_bfloat16*>(retTensor.data_),reinterpret_cast<__hip_bfloat16*>(t.data_), batch, channels, height, width, outH, outW,
      kernel.h, kernel.w, stride.h, stride.w, padding.h, padding.w, imStride,
      colW);
    }
  CUDA_KERNEL_CHECK();
  return retTensor;
}

TensorImpl TensorOpsCUDA::dot(const TensorImpl& a, const TensorImpl& b) {
  auto ret = TensorImpl::scalar(0.f, a.device_);
  auto sharedMemSize = getBlockSize() * sizeof(float);
  kDot<<<getGridSize(a.elemCount_), getBlockSize(), sharedMemSize>>>(
      ret.data_, a.data_, b.data_, a.elemCount_);
  CUDA_KERNEL_CHECK();
  return ret;
}

TensorImpl TensorOpsCUDA::triangle(const TensorImpl& t, int32_t diagonal,
                                    bool lower) {
  auto ret = TensorImpl::shape(t.shape_, t.device_);
  const auto rows = t.shape_[0];
  const auto cols = t.shape_[1];

  dim3 blockSize(WARP_SIZE, WARP_SIZE);
  dim3 gridSize((cols + blockSize.x - 1) / blockSize.x,
                (rows + blockSize.y - 1) / blockSize.y);

  if (lower) {
    kTriangle<true>
        <<<gridSize, blockSize>>>(ret.data_, t.data_, rows, cols, diagonal);
  } else {
    kTriangle<false>
        <<<gridSize, blockSize>>>(ret.data_, t.data_, rows, cols, diagonal);
  }
  CUDA_KERNEL_CHECK();
  return ret;
}

void TensorOpsCUDA::gemm(float* c, const float* a, const float* b, int32_t m,
                         int32_t k, int32_t n, bool transA, bool transB,const  Dtype Ta,
                         Dtype Tc) {
  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  int lda = transA ? m : k;
  int ldb = transB ? k : n;
  int ldc = n;

  const float alpha = 1.f;
  const float beta = 0.f;
  if (Ta == Dtype::float32)
    CUBLAS_CHECK(hipblasSgemm(getCublasHandle(), opB, opA, n, m, k, &alpha, b, ldb,
                           a, lda, &beta, c, ldc));
  else{
    hipDataType dtypea;
    DTYPE_SWITCH(Ta, dtypea);
    hipDataType dtypec;
    DTYPE_SWITCH(Tc, dtypec);
      CUBLAS_CHECK(hipblasGemmEx(
        getCublasHandle(),
        opB, opA,
        n, m, k,
        &alpha,
        b, dtypea, ldb,
        a, dtypea, lda,
        &beta,
        c, dtypec, ldc,
        HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT
    ));
  }
}

std::pair<TensorImpl, TensorImpl> TensorOpsCUDA::split(
    const TensorImpl& input,
    int32_t split_size0,
    int32_t split_size1,
    int32_t dim)
{
  Shape input_shape = input.shape();

  std::vector<int32_t> output_shape0 = input.shape();
  std::vector<int32_t> output_shape1 = input.shape();
  output_shape0[dim] = split_size0;
  output_shape1[dim] = split_size1;
  auto ret0 = TensorImpl::shape(output_shape0, input.device_, input.type_);
  auto ret1 = TensorImpl::shape(output_shape1, input.device_, input.type_);

  int32_t threads_per_block = 256;
  int32_t total_elems = input.numel();
  int32_t blocks = (total_elems + threads_per_block - 1) / threads_per_block;

  if (input.type() ==  Dtype::float32)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
        input.data(),
        ret0.data_,
        ret1.data_,
        input.shape().data(),
        split_size0,
        split_size1, dim,total_elems
    );
  else if (input.type() ==  Dtype::float16)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
       reinterpret_cast<const half*>(input.data()),
       reinterpret_cast<half*>(ret0.data_),
       reinterpret_cast<half*>(ret1.data_),
       input.shape().data(),
        split_size0,
        split_size1,
       dim, total_elems
   );
  else if (input.type() ==  Dtype::bfloat16)
    ppl_cukernel_split<<<blocks, threads_per_block>>>(
       reinterpret_cast<const __hip_bfloat16*>(input.data()),
       reinterpret_cast<__hip_bfloat16*>(ret0.data()),
       reinterpret_cast<__hip_bfloat16*>(ret1.data()),
       input.shape().data(),
        split_size0,
        split_size1,
       dim, total_elems
   );
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return {ret0, ret1};
}

TensorImpl TensorOpsCUDA::concat(const TensorImpl& a , const TensorImpl& b, int32_t dim){
  Shape a_shape = a.shape();
  Shape b_shape = b.shape();
  Shape output_shape = a_shape;
  output_shape[dim] = a_shape[dim] + b_shape[dim];

  TensorImpl ret = TensorImpl::shape(output_shape, a.device(), a.type());
  if (dim==a_shape.size()-1){
    size_t num_blocks = 1;
    size_t a_block_bytes = a_shape[dim] * sizeof(float);
    size_t b_block_bytes = b_shape[dim] * sizeof(float);
    size_t output_block_bytes = (a_shape[dim] + b_shape[dim]) * sizeof(float);

    for (int i = 0; i < a_shape.size() - 1; ++i) {
        num_blocks *= a_shape[i];
    }
    for (size_t i = 0; i < num_blocks; ++i) {
        const float* a_src = a.data() + i * a_shape[dim];
        const float* b_src = b.data() + i * b_shape[dim];
        float* output_dst = ret.data_ + i * (a_shape[dim] + b_shape[dim]);
        hipMemcpyAsync(
            output_dst,
            a_src,
            a_block_bytes,
            hipMemcpyDeviceToDevice
        );
        hipMemcpyAsync(
            output_dst + a_shape[dim],
            b_src,
            b_block_bytes,
            hipMemcpyDeviceToDevice
        );
    }
     return ret;
  }
  else{
     throw std::invalid_argument("Unsupported dim, we only support last dim concat");

  }
  return ret;
}

std::vector<TensorImpl> TensorOpsCUDA::concat_backward(const TensorImpl& grad, int32_t dim, int32_t a_dim_shape){
  Shape grad_shape = grad.shape();
  Shape output_shape_1 = grad.shape();
  Shape output_shape_2 = grad.shape();
  int32_t b_dim_shape = output_shape_2[dim] - a_dim_shape;
  output_shape_1[dim] = a_dim_shape;
  output_shape_2[dim] = b_dim_shape;

  TensorImpl ret0 = TensorImpl::shape(output_shape_1, grad.device(), grad.type());
  TensorImpl ret1 = TensorImpl::shape(output_shape_2, grad.device(), grad.type());
  if (dim==grad_shape.size()-1){
    const int64_t num_dims = output_shape_1.size();
    int64_t inner_size = 1;
    for (int i = 0; i < num_dims - 1; ++i) {

        inner_size *= output_shape_1[i];
    }

    const int64_t a_dim_size = output_shape_1[dim];
    const int64_t b_dim_size = output_shape_2[dim];
    const int64_t concat_dim_size = a_dim_size + b_dim_size;

    for (int64_t i = 0; i < inner_size; ++i) {
        const float* grad_output_ptr = grad.data() + i * concat_dim_size;
        float* grad_a_ptr = ret0.data_ + i * a_dim_size;
        float* grad_b_ptr = ret1.data_ + i * b_dim_size;
        hipMemcpyAsync(
            grad_a_ptr,
            grad_output_ptr,
            a_dim_size * sizeof(float),
            hipMemcpyDeviceToDevice
        );
        hipMemcpyAsync(
            grad_b_ptr,
            grad_output_ptr + a_dim_size,
            b_dim_size * sizeof(float),
            hipMemcpyDeviceToDevice
        );
    }
    }
  else{
      throw std::invalid_argument("Unsupported dim, we only support last dim concat");
  }
  return {ret0, ret1};
}

TensorImpl TensorOpsCUDA::upsample_forward(const TensorImpl& a , int32_t scale_factor){
  TensorImpl ret = TensorImpl::shape({a.shape_[0], a.shape_[1], static_cast<int>(a.shape_[2]*scale_factor),
                                            static_cast<int>(a.shape_[3]*scale_factor)}, a.device());
  int32_t N = a.numel();
  int32_t h = a.shape_[2];
  int32_t w = a.shape_[3];
  if (scale_factor == 2 && N >= 256){
    dim3 grid(N / kBlockSize, 1);
    dim3 block(kBlockSize, 1);
    UpsampleNearest2D2XForward<<<grid, block>>>(N, a.data_, h,
                                                w, ret.data_);
  }
  else{}
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return ret;
}

TensorImpl TensorOpsCUDA::upsample_backward(const TensorImpl& a , int32_t scale_factor){
  TensorImpl ret = TensorImpl::shape({a.shape_[0], a.shape_[1], static_cast<int>(a.shape_[2]/scale_factor),
                                            static_cast<int>(a.shape_[3]/scale_factor)}, a.device());
  int32_t N = a.numel();
  int32_t h = ret.shape_[2];
  int32_t w = ret.shape_[3];
  if (scale_factor == 2 && N >= 256){
    dim3 grid(N / kBlockSize, 1);
    dim3 block(kBlockSize, 1);
    UpsampleNearest2D2XBackward<<<grid, block>>>(N, a.data_, h,
                                                 w, ret.data_);
  }
  else
      {}
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel execution failed: %s\n", hipGetErrorString(err));
  }
  return ret;
}

const char* curandGetErrorString(hiprandStatus_t status) {
  switch (status) {
    case HIPRAND_STATUS_SUCCESS:
      return "HIPRAND_STATUS_SUCCESS";
    case HIPRAND_STATUS_VERSION_MISMATCH:
      return "HIPRAND_STATUS_VERSION_MISMATCH";
    case HIPRAND_STATUS_NOT_INITIALIZED:
      return "HIPRAND_STATUS_NOT_INITIALIZED";
    case HIPRAND_STATUS_ALLOCATION_FAILED:
      return "HIPRAND_STATUS_ALLOCATION_FAILED";
    case HIPRAND_STATUS_TYPE_ERROR:
      return "HIPRAND_STATUS_TYPE_ERROR";
    case HIPRAND_STATUS_OUT_OF_RANGE:
      return "HIPRAND_STATUS_OUT_OF_RANGE";
    case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
      return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
    case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
      return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
    case HIPRAND_STATUS_LAUNCH_FAILURE:
      return "HIPRAND_STATUS_LAUNCH_FAILURE";
    case HIPRAND_STATUS_PREEXISTING_FAILURE:
      return "HIPRAND_STATUS_PREEXISTING_FAILURE";
    case HIPRAND_STATUS_INITIALIZATION_FAILED:
      return "HIPRAND_STATUS_INITIALIZATION_FAILED";
    case HIPRAND_STATUS_ARCH_MISMATCH:
      return "HIPRAND_STATUS_ARCH_MISMATCH";
    case HIPRAND_STATUS_INTERNAL_ERROR:
      return "HIPRAND_STATUS_INTERNAL_ERROR";
  }
  return "Unknown cuRAND error";
}

const char* cublasGetErrorString(hipblasStatus_t status) {
  switch (status) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "Unknown cuBLAS error";
}

}  // namespace TinyTorch
